#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------------
 *
 *  Copyright (C) 1997-2010 Krzysztof M. Gorski, Eric Hivon,
 *                          Benjamin D. Wandelt, Anthony J. Banday,
 *                          Matthias Bartelmann,
 *                          Reza Ansari & Kenneth M. Ganga
 *
 *
 *  This file is part of HEALPix.
 *
 *  HEALPix is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  HEALPix is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with HEALPix; if not, write to the Free Software
 *  Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA  02110-1301  USA
 *
 *  For more information about HEALPix see http://healpix.jpl.nasa.gov
 *
 *----------------------------------------------------------------------------- */
/* ang2pix_ring.c
 *
 */

/* Standard Includes */
#include <cosmictiger/assert.hpp>
#include <cosmictiger/math.hpp>

__managed__ int x2pix[128], y2pix[128];

void healpix_init() {
	/* =======================================================================
	 * subroutine mk_xy2pix
	 * =======================================================================
	 * sets the array giving the number of the pixel lying in (x,y)
	 * x and y are in {1,128}
	 * the pixel number is in {0,128**2-1}
	 *
	 * if  i-1 = sum_p=0  b_p * 2^p
	 * then ix = sum_p=0  b_p * 4^p
	 * iy = 2*ix
	 * ix + iy in {0, 128**2 -1}
	 * =======================================================================
	 */
	int i, K, IP, I, J, ID;

	for (i = 0; i < 127; i++) {
		x2pix[i] = 0;
	}
	for (I = 1; I <= 128; I++) {
		J = I - 1; //            !pixel numbers
		K = 0; //
		IP = 1; //
		truc: if (J == 0) {
			x2pix[I - 1] = K;
			y2pix[I - 1] = 2 * K;
		} else {
			ID = J % 2;
			J = J / 2;
			K = IP * ID + K;
			IP = IP * 4;
			goto truc;
		}
	}

}

__device__ void vec2pix_nest(const long nside, double *vec, long *ipix) {

	/* =======================================================================
	 * subroutine vec2pix_nest(nside, vec, ipix)
	 * =======================================================================
	 * gives the pixel number ipix (NESTED) corresponding to vector vec
	 *
	 * the computation is made to the highest resolution available (nside=8192)
	 * and then degraded to that required (by integer division)
	 * this doesn't cost more, and it makes sure that the treatement of round-off
	 * will be consistent for every resolution
	 * =======================================================================
	 */

	double z, za, z0, tt, tp, tmp, phi;
	int face_num, jp, jm;
	long ifp, ifm;
	int ix, iy, ix_low, ix_hi, iy_low, iy_hi, ipf, ntt;
	const double piover2 = 0.5 * M_PI, twopi = 2.0 * M_PI;
	constexpr int ns_max = 8192;

	if (nside < 1 || nside > ns_max) {
		ALWAYS_ASSERT(false);
	}
	z = vec[2] / sqrt(vec[0] * vec[0] + vec[1] * vec[1] + vec[2] * vec[2]);
	phi = 0.0;
	if (vec[0] != 0.0 || vec[1] != 0.0) {
		phi = atan2(vec[1], vec[0]); /* in ]-pi, pi] */
		if (phi < 0.0) {
			phi += twopi; /* in  [0, 2pi[ */
		}
	}

	za = fabs(z);
	z0 = 2. / 3.;
	tt = phi / piover2; /* in [0,4[ */

	if (za <= z0) { /* equatorial region */

		/* (the index of edge lines increase when the longitude=phi goes up) */
		jp = (int) floor(ns_max * (0.5 + tt - z * 0.75)); /* ascending edge line index */
		jm = (int) floor(ns_max * (0.5 + tt + z * 0.75)); /* descending edge line index */

		/* finds the face */
		ifp = jp / ns_max; /* in {0,4} */
		ifm = jm / ns_max;

		if (ifp == ifm) {
			face_num = (int) (ifp % 4) + 4; /* faces 4 to 7 */
		} else if (ifp < ifm) {
			face_num = (int) (ifp % 4); /* (half-)faces 0 to 3 */
		} else {
			face_num = (int) (ifm % 4) + 8; /* (half-)faces 8 to 11 */
		}

		ix = (int) (jm % ns_max);
		iy = ns_max - (int) (jp % ns_max) - 1;
	} else { /* polar region, za > 2/3 */

		ntt = (int) floor(tt);
		if (ntt >= 4)
			ntt = 3;
		tp = tt - ntt;
		tmp = sqrt(3. * (1. - za)); /* in ]0,1] */

		/* (the index of edge lines increase when distance from the closest pole
		 * goes up)
		 */
		/* line going toward the pole as phi increases */
		jp = (int) floor(ns_max * tp * tmp);

		/* that one goes away of the closest pole */
		jm = (int) floor(ns_max * (1. - tp) * tmp);
		jp = (int) (jp < ns_max - 1 ? jp : ns_max - 1);
		jm = (int) (jm < ns_max - 1 ? jm : ns_max - 1);

		/* finds the face and pixel's (x,y) */
		if (z >= 0) {
			face_num = ntt; /* in {0,3} */
			ix = ns_max - jm - 1;
			iy = ns_max - jp - 1;
		} else {
			face_num = ntt + 8; /* in {8,11} */
			ix = jp;
			iy = jm;
		}
	}

	ix_low = (int) (ix % 128);
	ix_hi = ix / 128;
	iy_low = (int) (iy % 128);
	iy_hi = iy / 128;

	ipf = (x2pix[ix_hi] + y2pix[iy_hi]) * (128 * 128) + (x2pix[ix_low] + y2pix[iy_low]);
	ipf = (long) (ipf / sqr(ns_max / nside)); /* in {0, nside**2 - 1} */
	*ipix = (long) (ipf + face_num * sqr(nside)); /* in {0, 12*nside**2 - 1} */
}

