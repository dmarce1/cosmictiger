#include "hip/hip_runtime.h"
/*
 CosmicTiger - A cosmological N-Body code
 Copyright (C) 2021  Dominic C. Marcello

 This program is free software; you can redistribute it and/or
 modify it under the terms of the GNU General Public License
 as published by the Free Software Foundation; either version 2
 of the License, or (at your option) any later version.

 This program is distribufted in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program; if not, write to the Free Software
 Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 */

#include <cosmictiger/sph_cuda.hpp>

struct prehydro_record1 {
	fixed32 x;
	fixed32 y;
	fixed32 z;
	float h;
};

struct prehydro_record2 {
	float entr;
	float cfrac;
	float vx;
	float vy;
	float vz;
};

struct prehydro_workspace {
	device_vector<prehydro_record1> rec1;
	device_vector<prehydro_record2> rec2;
	device_vector<int> neighbors;
};

__global__ void sph_cuda_prehydro(sph_run_params params, sph_run_cuda_data data, sph_reduction* reduce) {
	const int tid = threadIdx.x;
	const int block_size = blockDim.x;
	__shared__
	int index;
	__shared__ prehydro_workspace ws;
	__syncthreads();
	if (tid == 0) {
		index = atomicAdd(&reduce->counter, 1);
	}
	__syncthreads();
	new (&ws) prehydro_workspace();
	array<fixed32, NDIM> x;

	const float gamma0 = data.def_gamma;
	const float code_to_energy = sqr(params.code_to_cm) / sqr(params.code_to_s);									// 5
	const float code_to_density = params.code_to_g / pow(params.code_to_cm, 3.);									// 12
	const float colog0 = log(1.5 * pow(constants::kb, 1.5) * pow(constants::e, -3) * pow(M_PI, -0.5));    // 35
	const float kappa0 = 20.0 * pow(2.0 / M_PI, 1.5) * pow(constants::kb, 2.5) * pow(constants::me, -0.5) * pow(constants::e, -4.0) * params.code_to_s
			* params.code_to_cm / (params.code_to_g * constants::avo);													// 49
	const float propc0 = 0.4 * (gamma0 - 1.0) * sqrtf(2.0 * constants::kb / (M_PI * constants::me)) / constants::c; // 17
	const float cv0 = constants::kb / (gamma0 - 1.0f);																		// 5
	const float invgm1 = 1.f / (gamma0 - 1.0);
	const float c0 = float(3.0f / (4.0f * M_PI)) * data.N;					// 1

	while (index < data.nselfs) {

		int flops = 0;
		__syncthreads();
		ws.rec1.resize(0);
		ws.rec2.resize(0);
		const sph_tree_node& self = data.trees[data.selfs[index]];
		bool found_self = false;
		for (int ni = self.neighbor_range.first; ni < self.neighbor_range.second; ni++) {
			const sph_tree_node& other = data.trees[data.neighbors[ni]];
			if (data.neighbors[ni] == data.selfs[index]) {
				found_self = true;
			}
			const int maxpi = round_up(other.part_range.second - other.part_range.first, block_size) + other.part_range.first;
			for (int pi = other.part_range.first + tid; pi < maxpi; pi += block_size) {
				bool contains = false;
				int j;
				int total;
				if (pi < other.part_range.second) {
					x[XDIM] = data.x[pi];
					x[YDIM] = data.y[pi];
					x[ZDIM] = data.z[pi];
					const float h_i = data.h[pi];
					if (self.outer_box.contains(x)) {
						contains = true;
					}
					if (!contains) {
						contains = true;
						for (int dim = 0; dim < NDIM; dim++) {
							if (distance(x[dim], self.inner_box.begin[dim]) + h_i < 0.f) {
								contains = false;
								flops += 3;
								break;
							}
							if (distance(self.inner_box.end[dim], x[dim]) + h_i < 0.f) {
								contains = false;
								flops += 3;
								break;
							}
						}
					}
				}
				j = contains;
				compute_indices < PREHYDRO_BLOCK_SIZE > (j, total);
				const int offset = ws.rec1.size();
				__syncthreads();
				const int next_size = offset + total;
				ws.rec1.resize(next_size);
				ws.rec2.resize(next_size);
				if (contains) {
					const int k = offset + j;
					ws.rec1[k].x = x[XDIM];
					ws.rec1[k].y = x[YDIM];
					ws.rec1[k].z = x[ZDIM];
					ws.rec1[k].h = data.h[pi];
					ws.rec2[k].vx = data.vx[pi];
					ws.rec2[k].vy = data.vy[pi];
					ws.rec2[k].vz = data.vz[pi];
					ws.rec2[k].entr = data.entr[pi];
					if (params.stars) {
						ws.rec2[k].cfrac = data.cold_frac[pi];
					} else {
						ws.rec2[k].cfrac = 0.f;
					}
				}
			}
		}
		ALWAYS_ASSERT(found_self);
		ALWAYS_ASSERT(ws.rec1.size());
		const float gamma0 = data.def_gamma;
		for (int i = self.part_range.first; i < self.part_range.second; i++) {
			__syncthreads();
			const int snki = self.sink_part_range.first - self.part_range.first + i;
			const bool active = data.rungs_snk[data.dm_index_snk[snki]] >= params.min_rung;
			x[XDIM] = data.x[i];
			x[YDIM] = data.y[i];
			x[ZDIM] = data.z[i];
			const fixed32& x_i = x[XDIM];
			const fixed32& y_i = x[YDIM];
			const fixed32& z_i = x[ZDIM];
			const float& h_i = data.rec2_snk[snki].h;
			const float hinv_i = 1.f / h_i; 										// 4
			const float h2_i = sqr(h_i);    										// 1
			int semiactive = 0;
			if (active) {
				if (tid == 0) {
					data.sa_snk[snki] = true;
				}
			} else {
				const int jmax = round_up(ws.rec1.size(), block_size);
				if (tid == 0) {
					data.sa_snk[snki] = false;
				}
				for (int j = tid; j < jmax; j += block_size) {
					if (j < ws.rec1.size()) {
						const auto x_j = ws.rec1[j].x;
						const auto y_j = ws.rec1[j].y;
						const auto z_j = ws.rec1[j].z;
						const auto h_j = ws.rec1[j].h;
						const auto h2_j = sqr(h_j);									// 1
						const float x_ij = distance(x_i, x_j);						// 1
						const float y_ij = distance(y_i, y_j);						// 1
						const float z_ij = distance(z_i, z_j);						// 1
						const float r2 = sqr(x_ij, y_ij, z_ij);					// 5
						if (r2 < fmaxf(h2_i, h2_j)) {									// 2
							semiactive++;
						}
						flops += 11;
					}
					shared_reduce_add<int, PREHYDRO_BLOCK_SIZE> (semiactive);
					if (semiactive) {
						break;
					}
				}
			}
			__syncthreads();
			if (active || semiactive) {
				float drho_dh;
				const float vx_i = data.vx[i];
				const float vy_i = data.vy[i];
				const float vz_i = data.vz[i];
				drho_dh = 0.f;
				float rhoh30 = (3.0f * data.N) / (4.0f * float(M_PI));   // 5
				const fixed32& x_i = x[XDIM];
				const fixed32& y_i = x[YDIM];
				const fixed32& z_i = x[ZDIM];
				float dvx_dx = 0.f;
				float dvx_dy = 0.f;
				float dvx_dz = 0.f;
				float dvy_dx = 0.f;
				float dvy_dy = 0.f;
				float dvy_dz = 0.f;
				float dvz_dx = 0.f;
				float dvz_dy = 0.f;
				float dvz_dz = 0.f;
				float pre = 0.f;
				float dpdh = 0.f;
				ws.neighbors.resize(0);
				__syncthreads();
				flops += 10;
				const int jmax = round_up(ws.rec1.size(), PREHYDRO_BLOCK_SIZE);
				for (int j = tid; j < jmax; j += block_size) {
					bool contains = false;
					if (j < ws.rec1.size()) {
						const auto& rec1 = ws.rec1[j];
						const fixed32& x_j = rec1.x;
						const fixed32& y_j = rec1.y;
						const fixed32& z_j = rec1.z;
						const float x_ij = distance(x_i, x_j); // 1
						const float y_ij = distance(y_i, y_j); // 1
						const float z_ij = distance(z_i, z_j); // 1
						const float r2 = sqr(x_ij, y_ij, z_ij);
						const float r = sqrtf(r2);                    // 4
						const float q = r * hinv_i;                    // 1
						if (q < 1.f) {                               // 1
							float w;
							const float dwdq = dkernelW_dq(q, &w, &flops);
							drho_dh -= q * dwdq;                      // 2
							contains = true;
							flops += 2;
						}
						flops += 9;
					}
					int k = contains;
					int total;
					compute_indices < PREHYDRO_BLOCK_SIZE > (k, total);
					const int offset = ws.neighbors.size();
					__syncthreads();
					const int next_size = offset + total;
					ws.neighbors.resize(next_size);
					if (contains) {
						const int l = offset + k;
						ws.neighbors[l] = j;
					}
				}
				shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(drho_dh);
				flops += (PREHYDRO_BLOCK_SIZE - 1);
				const float& m = data.m;
				drho_dh *= 0.33333333333f / rhoh30;								// 5
				const float fpre = 1.0f / drho_dh;								// 4
				__syncthreads();
				const float h3inv_i = hinv_i * sqr(hinv_i);					// 2
				const float h4inv_i = h3inv_i * hinv_i;						// 1
				const float rho_i = m * c0 * h3inv_i;
				float cfrac_i;
				if (params.stars) {
					cfrac_i = data.cold_frac[i];
				} else {
					cfrac_i = 0.f;
				}
				const float hfrac_i = 1.f - cfrac_i;									// 1
				const float& A_i = data.rec2_snk[snki].A;
				const float ene_i = A_i * powf(rho_i * hfrac_i, gamma0 - 1.0f);		// 11
				flops += 16;
				float gradx = 0.f;
				float grady = 0.f;
				float gradz = 0.f;
				for (int j = tid; j < ws.neighbors.size(); j += block_size) {
					const int kk = ws.neighbors[j];
					const auto& rec1 = ws.rec1[kk];
					const auto& rec2 = ws.rec2[kk];
					const fixed32& x_j = rec1.x;
					const fixed32& y_j = rec1.y;
					const fixed32& z_j = rec1.z;
					const float& h_j = rec1.h;
					const float& vx_j = rec2.vx;
					const float& vy_j = rec2.vy;
					const float& vz_j = rec2.vz;
					const float& A_j = rec2.entr;
					const float& fc_j = rec2.cfrac;
					const float fh_j = 1.f - fc_j;
					const float x_ij = distance(x_i, x_j);                  // 1
					const float y_ij = distance(y_i, y_j);                  // 1
					const float z_ij = distance(z_i, z_j);                  // 1
					const float r2 = sqr(x_ij, y_ij, z_ij);                 // 5
					const float r = sqrtf(r2);                               // 4
					const float q = r * hinv_i;                               // 1
					const float vx_ij = vx_i - vx_j + x_ij * params.adot;   // 3
					const float vy_ij = vy_i - vy_j + y_ij * params.adot;   // 3
					const float vz_ij = vz_i - vz_j + z_ij * params.adot;   // 3
					const float rinv = 1.0f / (1.0e-30f + r);               // 5
					float w;
					const float dwdq = dkernelW_dq(q, &w, &flops);
					const float dWdr_i = fpre * dwdq * h4inv_i;             // 2
					const float A0_j = fh_j * powf(A_j, 1.0f / gamma0);     // 9
					pre = fmaf(m, A0_j * w * h3inv_i, pre);                 // 4
					dpdh -= A0_j * (3.f * w + q * dwdq);                    // 5
					const float dWdr_i_rinv = dWdr_i * rinv;                // 1
					const float dWdr_i_x = dWdr_i_rinv * x_ij;				  // 1
					const float dWdr_i_y = dWdr_i_rinv * y_ij;              // 1
					const float dWdr_i_z = dWdr_i_rinv * z_ij;              // 1
					dvx_dx -= vx_ij * dWdr_i_x; // 2
					dvy_dx -= vy_ij * dWdr_i_x; // 2
					dvz_dx -= vz_ij * dWdr_i_x; // 2
					dvx_dy -= vx_ij * dWdr_i_y; // 2
					dvy_dy -= vy_ij * dWdr_i_y; // 2
					dvz_dy -= vz_ij * dWdr_i_y; // 2
					dvx_dz -= vx_ij * dWdr_i_z; // 2
					dvy_dz -= vy_ij * dWdr_i_z; // 2
					dvz_dz -= vz_ij * dWdr_i_z; // 2
					if (params.conduction) {
						const float hinv_j = 1.f / h_j;
						const float h3inv_j = sqr(hinv_j) * hinv_j;	// 2
						const float rho_j = m * c0 * h3inv_j;			// 2
						const float ene_j = A_j * powf(rho_j * fh_j, gamma0 - 1.0f); // 11
						const float tmp = dWdr_i  * logf(ene_j / ene_i); // 14
						gradx = fmaf(tmp, x_ij, gradx);					// 2
						grady = fmaf(tmp, y_ij, grady);					// 2
						gradz = fmaf(tmp, z_ij, gradz);					// 2
					}
					flops += 68;
				}
				if (params.conduction) {
					shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(gradx); //31
					shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(grady); //31
					shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(gradz); //31
					for (int j = tid; j < ws.neighbors.size(); j += block_size) {
						const auto& frac_i = data.rec1_snk[snki].frac;
						const float& cfrac_i = data.cold_mass_snk[snki];
						const float& H = frac_i[CHEM_H];
						const float& Hp = frac_i[CHEM_HP];
						const float& Hn = frac_i[CHEM_HN];
						const float& H2 = frac_i[CHEM_H2];
						const float& He = frac_i[CHEM_HE];
						const float& Hep = frac_i[CHEM_HEP];
						const float& Hepp = frac_i[CHEM_HEPP];
						const float grad2 = sqr(gradx, grady, gradz);	// 5
						const float gradToT = sqrtf(grad2);					// 4
						const float hfrac_i = 1.f - cfrac_i;				// 1
						const float rho0 = rho_i * hfrac_i / (sqr(params.a) * params.a); // 7
						float n0 = (H + fmaf(2.f, Hp, fmaf(.5f, H2, fmaf(.25f, He, fmaf(.5f, Hep, .75f * Hepp))))); // 10
						const float mmw_i = 1.0f / n0;						// 4
						const float ne_i = fmaxf((Hp - Hn + fmaf(0.25f, Hep, 0.5f * Hepp)) * rho0 * (constants::avo * code_to_density), 1e-30f);						// 8
						const float eint = code_to_energy * A_i * powf(rho0 * hfrac_i, gamma0 - 1.0) * invgm1; // 13
						const float T_i = mmw_i * eint / (cv0 * constants::avo); // 6
						const float colog_i = colog0 + 1.5f * logf(T_i) - 0.5f * logf(ne_i); // 20
						float kappa_i = (gamma0 - 1.f) * kappa0 * powf(T_i, 2.5f) / colog_i; // 15
						const float sigmax_i = propc0 * sqrtf(T_i);      // 5
						const float R = 2.f * mmw_i * kappa_i * gradToT / (rho_i * sigmax_i); // 7
						const float phi = (2.f + 3.f * R) / (2.f + 3.f * R + 3.f * sqr(R)); // 11
						kappa_i *= phi;											 // 1
						ALWAYS_ASSERT(isfinite(kappa_i));
						data.kap_snk[snki] = kappa_i;
						data.entr0_snk[snki] = A_i;
					}
					flops += 68;
				}
				shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(dpdh);       // 127
				shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(pre);			// 127
				shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(dvx_dx);		// 127
				shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(dvx_dy);		// 127
				shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(dvx_dz);		// 127
				shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(dvy_dx);		// 127
				shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(dvy_dy);		// 127
				shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(dvy_dz);		// 127
				shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(dvz_dx);		// 127
				shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(dvz_dy);		// 127
				shared_reduce_add<float, PREHYDRO_BLOCK_SIZE>(dvz_dz);		// 127
				if (tid == 0) {
					flops += 38 + (PREHYDRO_BLOCK_SIZE - 1) * 11;
					float shear_xx, shear_xy, shear_xz, shear_yy, shear_yz, shear_zz;
					float div_v;
					const float c0 = float(3.0f / 4.0f / M_PI * data.N);     // 1
					const float rho_i = c0 * h3inv_i;                        // 1
					const float mrhoinv = 1.f / rho_i;                       // 4
					dvx_dx *= mrhoinv;                                       // 1
					dvx_dy *= mrhoinv;                                       // 1
					dvx_dz *= mrhoinv;                                       // 1
					dvy_dx *= mrhoinv;                                       // 1
					dvy_dy *= mrhoinv;                                       // 1
					dvy_dz *= mrhoinv;                                       // 1
					dvz_dx *= mrhoinv;                                       // 1
					dvz_dy *= mrhoinv;                                       // 1
					dvz_dz *= mrhoinv;                                       // 1
					pre = powf(pre, gamma0);                                 // 4
					dpdh *= 0.33333333333f / rhoh30;                         // 5
					div_v = dvx_dx + dvy_dy + dvz_dz;                        // 2
					shear_xx = dvx_dx - (1.f / 3.f) * div_v;                 // 2
					shear_yy = dvy_dy - (1.f / 3.f) * div_v;                 // 2
					shear_zz = dvz_dz - (1.f / 3.f) * div_v;                 // 2
					shear_xy = 0.5f * (dvx_dy + dvy_dx);                     // 2
					shear_xz = 0.5f * (dvx_dz + dvz_dx);                     // 2
					shear_yz = 0.5f * (dvy_dz + dvz_dy);                     // 2
					const float shearv = sqrtf(sqr(shear_xx) + sqr(shear_yy) + sqr(shear_zz) + 2.0f * (sqr(shear_xy) + sqr(shear_xz) + sqr(shear_yz))); // 16
					data.shear_snk[snki] = shearv;
					data.fpre1_snk[snki] = fpre;
					data.fpre2_snk[snki] = dpdh;
					data.pre_snk[snki] = pre;
				}
			}
		}
		shared_reduce_add<int, PREHYDRO_BLOCK_SIZE>(flops);
		if (tid == 0) {
			atomicAdd(&reduce->flops, (double) flops);
			index = atomicAdd(&reduce->counter, 1);
		}
		flops = 0;
		__syncthreads();
	}
	(&ws)->~prehydro_workspace();
}

