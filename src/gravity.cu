#include "hip/hip_runtime.h"
/*
 CosmicTiger - A cosmological N-Body code
 Copyright (C) 2021  Dominic C. Marcello

 This program is free software; you can redistribute it and/or
 modify it under the terms of the GNU General Public License
 as published by the Free Software Foundation; either version 2
 of the License, or (at your option) any later version.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program; if not, write to the Free Software
 Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 */

#include <cosmictiger/cuda_reduce.hpp>
#include <cosmictiger/gravity.hpp>
#include <cosmictiger/kernel.hpp>

__device__
int cuda_gravity_cc(const cuda_kick_data& data, expansion<float>& Lacc, const tree_node& self, const fixedcapvec<int, MULTLIST_SIZE>& multlist,
		gravity_cc_type type, bool do_phi) {
	int flops = 0;
	const int &tid = threadIdx.x;
	const auto& tree_nodes = data.tree_nodes;
	if (multlist.size()) {
		expansion<float> L;
		expansion<float> D;
		for (int i = 0; i < EXPANSION_SIZE; i++) {
			L[i] = 0.0f;
		}
		for (int i = tid; i < multlist.size(); i += WARP_SIZE) {
			const tree_node& other = tree_nodes[multlist[i]];
			const multipole<float>& M = other.multi;
			array<float, NDIM> dx;
			for (int dim = 0; dim < NDIM; dim++) {
				dx[dim] = distance(self.pos[dim], other.pos[dim]);
			}
			flops += 3;
			if (type == GRAVITY_CC_DIRECT) {
				flops += greens_function(D, dx);
			} else {
				flops += ewald_greens_function(D, dx);
			}
			flops += M2L(L, M, D, do_phi);
		}
		for (int i = 0; i < EXPANSION_SIZE; i++) {
			shared_reduce_add(L[i]);
		}
		for (int i = tid; i < EXPANSION_SIZE; i += WARP_SIZE) {
			Lacc[i] += L[i];
		}
		__syncwarp();
	}
	shared_reduce_add(flops);
	return flops;
}

__device__
int cuda_gravity_cp(const cuda_kick_data& data, expansion<float>& Lacc, const tree_node& self, const fixedcapvec<int, PARTLIST_SIZE>& partlist, bool do_phi) {
	int flops = 0;
	__shared__
	extern int shmem_ptr[];
	const bool sph = data.src_mass != nullptr;
	cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
	const auto* main_src_x = data.x;
	const auto* main_src_y = data.y;
	const auto* main_src_z = data.z;
	const auto* main_src_mass = data.src_mass;
	auto& src_x = shmem.src.x;
	auto& src_y = shmem.src.y;
	auto& src_z = shmem.src.z;
	auto& src_mass = shmem.src.mass;
	const auto* tree_nodes = data.tree_nodes;
	const int &tid = threadIdx.x;
	if (partlist.size()) {
		int part_index;
		expansion<float> L;
		for (int j = 0; j < EXPANSION_SIZE; j++) {
			L[j] = 0.0;
		}
		int i = 0;
		auto these_parts = tree_nodes[partlist[0]].part_range;
		const auto partsz = partlist.size();
		while (i < partsz) {
			part_index = 0;
			while (part_index < KICK_PP_MAX && i < partsz) {
				while (i + 1 < partsz) {
					const auto other_tree_parts = tree_nodes[partlist[i + 1]].part_range;
					if (these_parts.second == other_tree_parts.first) {
						these_parts.second = other_tree_parts.second;
						i++;
					} else {
						break;
					}
				}
				const part_int imin = these_parts.first;
				const part_int imax = min(these_parts.first + (KICK_PP_MAX - part_index), these_parts.second);
				const int sz = imax - imin;
				for (int j = tid; j < sz; j += WARP_SIZE) {
					const int i1 = part_index + j;
					const part_int i2 = j + imin;
					ASSERT(i2 >= 0);
					ASSERT(i2 < data.source_size);
					src_x[i1] = main_src_x[i2];
					src_y[i1] = main_src_y[i2];
					src_z[i1] = main_src_z[i2];
					if (sph) {
						src_mass[i1] = main_src_mass[i2];
					}
				}
				__syncwarp();
				these_parts.first += sz;
				part_index += sz;
				if (these_parts.first == these_parts.second) {
					i++;
					if (i < partsz) {
						these_parts = tree_nodes[partlist[i]].part_range;
					}
				}
			}
			__syncwarp();
			for (int j = tid; j < part_index; j += warpSize) {
				array<float, NDIM> dx;
				dx[XDIM] = distance(self.pos[XDIM], src_x[j]);
				dx[YDIM] = distance(self.pos[YDIM], src_y[j]);
				dx[ZDIM] = distance(self.pos[ZDIM], src_z[j]);
				float mass = 1.f;
				if (data.src_mass) {
					mass = src_mass[j];
				}
				flops += 3;
				expansion<float> D;
				flops += greens_function(D, dx);
				for (int k = 0; k < EXPANSION_SIZE; k++) {
					L[k] += mass * D[k];
				}
				flops += 2 * EXPANSION_SIZE;
			}
		}
		for (int k = 0; k < EXPANSION_SIZE; k++) {
			shared_reduce_add(L[k]);
		}
		for (int i = tid; i < EXPANSION_SIZE; i += WARP_SIZE) {
			Lacc[i] += L[i];
		}

		__syncwarp();
	}
	shared_reduce_add(flops);
	return flops;

}

__device__
int cuda_gravity_pc(const cuda_kick_data& data, const tree_node&, const fixedcapvec<int, MULTLIST_SIZE>& multlist, int nactive, bool do_phi) {
	int flops = 0;
	const int &tid = threadIdx.x;
	__shared__
	extern int shmem_ptr[];
	cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
	auto &gx = shmem.gx;
	auto &gy = shmem.gy;
	auto &gz = shmem.gz;
	auto &phi = shmem.phi;
	const auto& sink_x = shmem.sink_x;
	const auto& sink_y = shmem.sink_y;
	const auto& sink_z = shmem.sink_z;
	const auto* tree_nodes = data.tree_nodes;
	if (multlist.size()) {
		__syncwarp();
		for (int k = tid; k < nactive; k += WARP_SIZE) {
			expansion2<float> L;
			L(0, 0, 0) = L(1, 0, 0) = L(0, 1, 0) = L(0, 0, 1) = 0.0f;
			for (int j = 0; j < multlist.size(); j++) {
				array<float, NDIM> dx;
				const auto& pos = tree_nodes[multlist[j]].pos;
				const auto& M = tree_nodes[multlist[j]].multi;
				dx[XDIM] = distance(sink_x[k], pos[XDIM]);
				dx[YDIM] = distance(sink_y[k], pos[YDIM]);
				dx[ZDIM] = distance(sink_z[k], pos[ZDIM]);
				flops += 3;
				expansion<float> D;
				flops += greens_function(D, dx);
				flops += M2L(L, M, D, do_phi);
			}
			gx[k] -= L(1, 0, 0);
			gy[k] -= L(0, 1, 0);
			gz[k] -= L(0, 0, 1);
			phi[k] += L(0, 0, 0);
		}
		__syncwarp();
	}
	__syncwarp();
	shared_reduce_add(flops);
	__syncwarp();
	return flops;

}

__device__
int cuda_gravity_pp(const cuda_kick_data& data, const tree_node& self, const fixedcapvec<int, PARTLIST_SIZE>& partlist, int nactive, float h, bool do_phi) {
	const int &tid = threadIdx.x;
	const bool sph = data.src_mass != nullptr;
	__shared__
	extern int shmem_ptr[];
	cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
	auto &gx = shmem.gx;
	auto &gy = shmem.gy;
	auto &gz = shmem.gz;
	auto &phi = shmem.phi;
	const bool vsoft = sph && data.vsoft;
	auto& src_hsoft = shmem.src.hsoft;
	const auto& sink_hsoft = shmem.sink_hsoft;
	const auto& sink_x = shmem.sink_x;
	const auto& sink_y = shmem.sink_y;
	const auto& sink_z = shmem.sink_z;
	const auto* main_src_x = data.x;
	const auto* main_src_y = data.y;
	const auto* main_src_z = data.z;
	const auto* main_src_mass = data.src_mass;
	const auto* main_src_hsoft = data.hsoft;
	auto& src_x = shmem.src.x;
	auto& src_y = shmem.src.y;
	auto& src_z = shmem.src.z;
	auto& src_mass = shmem.src.mass;
	const auto* tree_nodes = data.tree_nodes;
	float h2 = sqr(h);
	float hinv = 1.f / (h);
	float h3inv = hinv * hinv * hinv;
	int part_index;
	int nnear = 0;
	int nfar = 0;
	int flops = 0;
	if (partlist.size()) {
		int i = 0;
		auto these_parts = tree_nodes[partlist[0]].part_range;
		const auto partsz = partlist.size();
		while (i < partsz) {
			part_index = 0;
			while (part_index < KICK_PP_MAX && i < partsz) {
				while (i + 1 < partsz) {
					const auto other_tree_parts = tree_nodes[partlist[i + 1]].part_range;
					if (these_parts.second == other_tree_parts.first) {
						these_parts.second = other_tree_parts.second;
						i++;
					} else {
						break;
					}
				}
				const part_int imin = these_parts.first;
				const part_int imax = min(these_parts.first + (KICK_PP_MAX - part_index), these_parts.second);
				const int sz = imax - imin;
				for (int j = tid; j < sz; j += WARP_SIZE) {
					const int i1 = part_index + j;
					const part_int i2 = j + imin;
					ASSERT(i2 >= 0);
					ASSERT(i2 < data.source_size);
					src_x[i1] = main_src_x[i2];
					src_y[i1] = main_src_y[i2];
					src_z[i1] = main_src_z[i2];
					if (sph) {
						src_mass[i1] = main_src_mass[i2];
						if (vsoft) {
							src_hsoft[i1] = main_src_hsoft[i2];
						}
					}
				}
				__syncwarp();
				these_parts.first += sz;
				part_index += sz;
				if (these_parts.first == these_parts.second) {
					i++;
					if (i < partsz) {
						these_parts = tree_nodes[partlist[i]].part_range;
					}
				}
			}
			float fx;
			float fy;
			float fz;
			float pot;
			float dx0;
			float dx1;
			float dx2;
			float r3inv;
			float r1inv;
			__syncwarp();
			if (sph) {
				for (int k = tid; k < nactive; k += WARP_SIZE) {
					fx = 0.f;
					fy = 0.f;
					fz = 0.f;
					pot = 0.f;
					for (int j = 0; j < part_index; j++) {
						dx0 = distance(sink_x[k], src_x[j]); // 1
						dx1 = distance(sink_y[k], src_y[j]); // 1
						dx2 = distance(sink_z[k], src_z[j]); // 1
						const float mass = src_mass[j];
						const auto r2 = sqr(dx0, dx1, dx2);  // 5
						if (vsoft) {
							h = max(sink_hsoft[k], src_hsoft[j]);
							h2 = sqr(h);
						}
						if (r2 >= h2) {                      // 1
							r1inv = rsqrt(r2);                // 4
							r3inv = r1inv * r1inv * r1inv;    // 2
							nnear++;
						} else {
							if (vsoft) {
								hinv = 1.0f / h;
								h3inv = hinv * sqr(hinv);
							}
							const float q = fminf(sqrt(r2) * hinv, 1.f);
							r3inv = kernelFqinv(q) * h3inv;
							if (do_phi) {
								r1inv = kernelPot(q) * hinv;
							}
							nfar++;
						}
						r3inv *= mass;
						r1inv *= mass;
						flops += 2;
						fx = fmaf(dx0, r3inv, fx);                     // 2
						fy = fmaf(dx1, r3inv, fy);                     // 2
						fz = fmaf(dx2, r3inv, fz);                     // 2
						pot -= r1inv;                                  // 1
					}
					gx[k] -= fx;
					gy[k] -= fy;
					gz[k] -= fz;
					phi[k] += pot;
				}
			} else {
				for (int k = tid; k < nactive; k += WARP_SIZE) {
					fx = 0.f;
					fy = 0.f;
					fz = 0.f;
					pot = 0.f;
					for (int j = 0; j < part_index; j++) {
						dx0 = distance(sink_x[k], src_x[j]); // 1
						dx1 = distance(sink_y[k], src_y[j]); // 1
						dx2 = distance(sink_z[k], src_z[j]); // 1
						const auto r2 = sqr(dx0, dx1, dx2);  // 5
						if (r2 >= h2) {                      // 1
							r1inv = rsqrt(r2);                // 4
							r3inv = r1inv * r1inv * r1inv;    // 2
							nnear++;
						} else {
							const float r1oh1 = sqrtf(r2) * hinv; // 5
							const float r2oh2 = r1oh1 * r1oh1;    // 1
							r3inv = +15.0f / 8.0f;
							r1inv = -5.0f / 16.0f;
							r3inv = fmaf(r3inv, r2oh2, -21.0f / 4.0f);  // 2
							r1inv = fmaf(r1inv, r2oh2, 21.0f / 16.0f);  // 2
							r3inv = fmaf(r3inv, r2oh2, +35.0f / 8.0f);  // 2
							r1inv = fmaf(r1inv, r2oh2, -35.0f / 16.0f); // 2
							r3inv *= h3inv;                             // 1
							r1inv = fmaf(r1inv, r2oh2, 35.0f / 16.0f);  // 2
							r1inv *= hinv;                              // 1
							nfar++;
						}
						fx = fmaf(dx0, r3inv, fx);                     // 2
						fy = fmaf(dx1, r3inv, fy);                     // 2
						fz = fmaf(dx2, r3inv, fz);                     // 2
						pot -= r1inv;                                  // 1
					}
					gx[k] -= fx;
					gy[k] -= fy;
					gz[k] -= fz;
					phi[k] += pot;
				}
			}
		}
		__syncwarp();
	}
	shared_reduce_add(nnear);
	shared_reduce_add(nfar);
	shared_reduce_add(flops);
	__syncwarp();
	return nfar * 22 + 37 * nnear + flops;

}
