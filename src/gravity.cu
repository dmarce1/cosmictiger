#include "hip/hip_runtime.h"
/*
 CosmicTiger - A cosmological N-Body code
 Copyright (C) 2021  Dominic C. Marcello

 This program is free software; you can redistribute it and/or
 modify it under the terms of the GNU General Public License
 as published by the Free Software Foundation; either version 2
 of the License, or (at your option) any later version.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program; if not, write to the Free Software
 Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 */

#include <cosmictiger/cuda_reduce.hpp>
#include <cosmictiger/flops.hpp>
#include <cosmictiger/gravity.hpp>
#include <cosmictiger/kernels.hpp>
#include <hip/hip_cooperative_groups.h>
#include <cuda/barrier>

__managed__ double iclose;
__managed__ double idirect;
__managed__ double inparts;
__managed__ bool use_gravity_counters = false;

void reset_gravity_counters() {
	iclose = idirect = inparts = 0;
}

void set_gravity_counter_use(bool code) {
	use_gravity_counters = code;
}

void get_gravity_counters(double& close, double& direct) {
	close = iclose / inparts;
	direct = idirect / inparts;
}

__device__
void cuda_gravity_cc_direct(const cuda_kick_data& data, expansion<float>& Lacc, const tree_node& self, const device_vector<int>& multlist, bool do_phi) {
	const int &tid = threadIdx.x;
	const auto& tree_nodes = data.tree_nodes;
	flop_counter<int> flops = 0;
	if (multlist.size()) {
		//	PRINT("CC\n");

#ifdef TREEPM
		const float rsinv = 1.f / data.rs;
		const float rsinv2 = sqr(rsinv);
#endif
		expansion<float> L;

		for (int i = 0; i < EXPANSION_SIZE; i++) {
			L[i] = 0.0f;
		}
		for (int i = tid; i < multlist.size(); i += WARP_SIZE) {
			const tree_node& other = tree_nodes[multlist[i]];
			const multipole<float>& M = other.mpos->multi;
			sfmm::vec3<float> dx;
			for (int dim = 0; dim < NDIM; dim++) {
				dx[dim] = distance(self.mpos->pos[dim], other.mpos->pos[dim]);
			}
			flops += M2L(L, M, dx, do_phi);
		}
		shared_reduce_add_array<float, EXPANSION_SIZE>(L.data());
		for (int i = tid; i < EXPANSION_SIZE; i += WARP_SIZE) {
			Lacc[i] += L[i];
		}
		flops += 6 * EXPANSION_SIZE;
		__syncwarp();
	}
	add_gpu_flops(flops);
}

__device__
void cuda_gravity_cp_direct(const cuda_kick_data& data, expansion<float>& Lacc, const tree_node& self, const device_vector<int>& partlist, bool do_phi) {
	__shared__
	extern int shmem_ptr[];
	cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
	const auto* main_src_x = data.x;
	const auto* main_src_y = data.y;
	const auto* main_src_z = data.z;
	auto& src_x = shmem.X.x;
	auto& src_y = shmem.X.y;
	auto& src_z = shmem.X.z;
	auto& barrier = shmem.barrier;
	const auto* tree_nodes = data.tree_nodes;
	const int &tid = threadIdx.x;
	flop_counter<int> flops = 0;
	if (partlist.size()) {
		//	PRINT("CP\n");

#ifdef TREEPM
		const float rsinv = 1.f / data.rs;
		const float rsinv2 = sqr(rsinv);
#endif
		int part_index;
		expansion<float> L;
		for (int j = 0; j < EXPANSION_SIZE; j++) {
			L[j] = 0.0;
		}
		int i = 0;
		auto these_parts = tree_nodes[partlist[0]].part_range;
		const auto partsz = partlist.size();
		while (i < partsz) {
			auto group = cooperative_groups::this_thread_block();
			if (group.thread_rank() == 0) {
				init(&barrier, group.size());
			}
			group.sync();
			part_index = 0;
			while (part_index < KICK_PP_MAX && i < partsz) {
				while (i + 1 < partsz) {
					const auto other_tree_parts = tree_nodes[partlist[i + 1]].part_range;
					if (these_parts.second == other_tree_parts.first) {
						these_parts.second = other_tree_parts.second;
						i++;
					} else {
						break;
					}
				}
				const part_int imin = these_parts.first;
				const part_int imax = min(these_parts.first + (KICK_PP_MAX - part_index), these_parts.second);
				const int sz = imax - imin;
				cuda::memcpy_async(group, src_x.data() + part_index, main_src_x + imin, sizeof(fixed32) * sz, barrier);
				cuda::memcpy_async(group, src_y.data() + part_index, main_src_y + imin, sizeof(fixed32) * sz, barrier);
				cuda::memcpy_async(group, src_z.data() + part_index, main_src_z + imin, sizeof(fixed32) * sz, barrier);
				__syncwarp();
				these_parts.first += sz;
				part_index += sz;
				if (these_parts.first == these_parts.second) {
					i++;
					if (i < partsz) {
						these_parts = tree_nodes[partlist[i]].part_range;
					}
				}
			}
			barrier.arrive_and_wait();
			__syncwarp();
			for (int j = tid; j < part_index; j += warpSize) {
				sfmm::vec3<float> dx;
				dx[XDIM] = distance(self.mpos->pos[XDIM], src_x[j]);
				dx[YDIM] = distance(self.mpos->pos[YDIM], src_y[j]);
				dx[ZDIM] = distance(self.mpos->pos[ZDIM], src_z[j]);
				P2L(L, 1.0f, dx, do_phi);
			}
		}
		shared_reduce_add_array<float, EXPANSION_SIZE>(L.data());
		for (int i = tid; i < EXPANSION_SIZE; i += WARP_SIZE) {
			Lacc[i] += L[i];
		}
		flops += 6 * EXPANSION_SIZE;

		__syncwarp();
	}
	add_gpu_flops(flops);
}

__device__
void cuda_gravity_pc_direct(const cuda_kick_data& data, const tree_node& self, const device_vector<int>& multlist, bool do_phi) {
	const int &tid = threadIdx.x;
	__shared__
	extern int shmem_ptr[];
	cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
	auto &force = shmem.f;
	auto& multis = shmem.mpos;
	const int nsink = self.part_range.second - self.part_range.first;
	const auto* sink_x = data.x + self.part_range.first;
	const auto* sink_y = data.y + self.part_range.first;
	const auto* sink_z = data.z + self.part_range.first;
	auto& barrier = shmem.barrier;
	const auto* tree_nodes = data.tree_nodes;
	flop_counter<int> flops = 0;
	auto group = cooperative_groups::this_thread_block();
	if (multlist.size()) {
		__syncwarp();
		int mi = 0;
		const int cnt = multlist.size();
		while (mi < cnt) {
			int mend = min(cnt, mi + KICK_C_MAX);
			for (int this_mi = mi; this_mi < mend; this_mi++) {
				cuda::memcpy_async(group, (void*) &multis[this_mi - mi], tree_nodes[multlist[this_mi]].mpos, sizeof(multi_pos), barrier);
			}
			mend -= mi;
			mi += mend;
			barrier.arrive_and_wait();
			force_type<float> f;
			sfmm::vec3<float> dx;
			for (int j = 0; j < mend; j++) {
				const auto& pos = multis[j].pos;
				const auto& M = multis[j].multi;
				for (int k = tid; k < nsink; k += WARP_SIZE) {
					auto& F = force[k];
					f.init();
					dx[XDIM] = distance(sink_x[k], pos[XDIM]);
					dx[YDIM] = distance(sink_y[k], pos[YDIM]);
					dx[ZDIM] = distance(sink_z[k], pos[ZDIM]);
					flops += M2P(f, M, dx, do_phi);
					F += f;
					flops += 4;
				}
			}
			__syncwarp();
		}
	}
	__syncwarp();
	add_gpu_flops(flops);
}

__device__
void cuda_gravity_pp_direct(const cuda_kick_data& data, const tree_node& self, const device_vector<int>& partlist, float h, bool do_phi) {
	const int &tid = threadIdx.x;
	__shared__
	extern int shmem_ptr[];
	cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
	auto &force = shmem.f;
	const int nsink = self.part_range.second - self.part_range.first;
	const auto* sink_x = data.x + self.part_range.first;
	const auto* sink_y = data.y + self.part_range.first;
	const auto* sink_z = data.z + self.part_range.first;
	auto& barrier = shmem.barrier;
	const auto* main_src_x = data.x;
	const auto* main_src_y = data.y;
	const auto* main_src_z = data.z;
	auto& src_x = shmem.X.x;
	auto& src_y = shmem.X.y;
	auto& src_z = shmem.X.z;
	const auto* tree_nodes = data.tree_nodes;
	int part_index;
	const float h2 = sqr(h);
	const float hinv = 1.f / h;
	const float h2inv = sqr(hinv);
	const float h3inv = h2inv * hinv;
	flop_counter<int> flops = 7;
	int close = 0;
	int direct = 0;
	if (partlist.size()) {
		int i = 0;
		auto these_parts = tree_nodes[partlist[0]].part_range;
		const auto partsz = partlist.size();
		while (i < partsz) {
			auto group = cooperative_groups::this_thread_block();
			part_index = 0;
			while (part_index < KICK_PP_MAX && i < partsz) {
				while (i + 1 < partsz) {
					const auto other_tree_parts = tree_nodes[partlist[i + 1]].part_range;
					if (these_parts.second == other_tree_parts.first) {
						these_parts.second = other_tree_parts.second;
						i++;
					} else {
						break;
					}
				}
				const part_int imin = these_parts.first;
				const part_int imax = min(these_parts.first + (KICK_PP_MAX - part_index), these_parts.second);
				const int sz = imax - imin;
				cuda::memcpy_async(group, src_x.data() + part_index, main_src_x + imin, sizeof(fixed32) * sz, barrier);
				cuda::memcpy_async(group, src_y.data() + part_index, main_src_y + imin, sizeof(fixed32) * sz, barrier);
				cuda::memcpy_async(group, src_z.data() + part_index, main_src_z + imin, sizeof(fixed32) * sz, barrier);
				__syncwarp();
				these_parts.first += sz;
				part_index += sz;
				if (these_parts.first == these_parts.second) {
					i++;
					if (i < partsz) {
						these_parts = tree_nodes[partlist[i]].part_range;
					}
				}
			}
			barrier.arrive_and_wait();
			float fx;
			float fy;
			float fz;
			float pot;
			float dx0;
			float dx1;
			float dx2;
			float f;
			float phi;
			__syncwarp();
			int kmid = (nsink / WARP_SIZE) * WARP_SIZE;
			if (nsink - kmid >= WARP_SIZE / 2) {
				kmid = nsink;
			}
			for (int k = tid; k < kmid; k += WARP_SIZE) {
				fx = 0.f;
				fy = 0.f;
				fz = 0.f;
				pot = 0.f;
				for (int j = 0; j < part_index; j++) {
					dx0 = distance(sink_x[k], src_x[j]); // 1
					dx1 = distance(sink_y[k], src_y[j]); // 1
					dx2 = distance(sink_z[k], src_z[j]); // 1
					const auto r2 = sqr(dx0, dx1, dx2);  // 5
					if (r2 > h2) {
						phi = rsqrt(r2);					// 4
						f = sqr(phi) * phi;					// 2
						direct++;
					} else {
						close++;
						gsoft(f, phi, r2, hinv, h2inv, h3inv, do_phi);
					}
					fx = fmaf(dx0, f, fx);                     // 2
					fy = fmaf(dx1, f, fy);                     // 2
					fz = fmaf(dx2, f, fz);                     // 2
					pot -= phi;                                  // 1
					flops += 21;
				}
				auto& F = force[k];
				F.force[0] -= fx;
				F.force[1] -= fy;
				F.force[2] -= fz;
				flops += 4;
				F.potential += pot;
			}
			for (int k = kmid; k < nsink; k++) {
				fx = 0.f;
				fy = 0.f;
				fz = 0.f;
				pot = 0.f;
				for (int j = tid; j < part_index; j += WARP_SIZE) {
					dx0 = distance(sink_x[k], src_x[j]); // 2
					dx1 = distance(sink_y[k], src_y[j]); // 2
					dx2 = distance(sink_z[k], src_z[j]); // 2
					const auto r2 = sqr(dx0, dx1, dx2);  // 5
					if (r2 > h2) {
						phi = rsqrt(r2);					// 4
						f = sqr(phi) * phi;					// 2
						direct++;
					} else {
						close++;
						gsoft(f, phi, r2, hinv, h2inv, h3inv, do_phi);
					}
					fx = fmaf(dx0, f, fx);                     // 2
					fy = fmaf(dx1, f, fy);                     // 2
					fz = fmaf(dx2, f, fz);                     // 2
					pot -= phi;                                  // 1
					flops += 23;
				}
				shared_reduce_add(fx);
				shared_reduce_add(fy);
				shared_reduce_add(fz);
				if (do_phi) {
					shared_reduce_add(pot);
				}
				if (tid == 0) {
					auto& F = force[k];
					F.force[0] -= fx;
					F.force[1] -= fy;
					F.force[2] -= fz;
					flops += 4;
					F.potential += pot;
				}
			}
		}
		__syncwarp();
	}
	shared_reduce_add(close);
	shared_reduce_add(direct);
	if (tid == 0 && use_gravity_counters) {
		atomicAdd(&iclose, close);
		atomicAdd(&idirect, direct);
		atomicAdd(&inparts, nsink);
	}

	__syncwarp();
	add_gpu_flops(flops);

}

#ifdef FMM

__device__
void cuda_gravity_cc_ewald(const cuda_kick_data& data, expansion<float>& Lacc, const tree_node& self, const device_vector<int>& multlist, bool do_phi) {
	const int &tid = threadIdx.x;
	const auto& tree_nodes = data.tree_nodes;
	flop_counter<int> flops = 0;
	if (multlist.size()) {
		expansion<float> L;

		for (int i = 0; i < EXPANSION_SIZE; i++) {
			L[i] = 0.0f;
		}
		for (int i = tid; i < multlist.size(); i += WARP_SIZE) {
			const tree_node& other = tree_nodes[multlist[i]];
			multipole<float> M = other.mpos->multi;
			//	flops += apply_scale_factor(M);
			sfmm::vec3<float> dx;
			for (int dim = 0; dim < NDIM; dim++) {
				dx[dim] = distance(self.mpos->pos[dim], other.mpos->pos[dim]);
			}
			flops += M2L_ewald(L, M, dx, do_phi);
		}
		//flops += apply_scale_factor_inv(L);
		shared_reduce_add_array<float, EXPANSION_SIZE>(L.data());
		for (int i = tid; i < EXPANSION_SIZE; i += WARP_SIZE) {
			Lacc[i] += L[i];
		}
		flops += 6 * EXPANSION_SIZE;
		__syncwarp();
	}
	add_gpu_flops(flops);
}

__device__
void cuda_gravity_cp_ewald(const cuda_kick_data& data, expansion<float>& Lacc, const tree_node& self, const device_vector<int>& partlist, bool do_phi) {
	__shared__
	extern int shmem_ptr[];
	cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
	const auto* main_src_x = data.x;
	const auto* main_src_y = data.y;
	const auto* main_src_z = data.z;
	auto& src_x = shmem.X.x;
	auto& src_y = shmem.X.y;
	auto& src_z = shmem.X.z;
	auto& barrier = shmem.barrier;
	const auto* tree_nodes = data.tree_nodes;
	const int &tid = threadIdx.x;
	flop_counter<int> flops = 0;
	if (partlist.size()) {
		int part_index;
		expansion<float> L;
		for (int j = 0; j < EXPANSION_SIZE; j++) {
			L[j] = 0.0;
		}
		int i = 0;
		auto these_parts = tree_nodes[partlist[0]].part_range;
		const auto partsz = partlist.size();
		while (i < partsz) {
			auto group = cooperative_groups::this_thread_block();
			if (group.thread_rank() == 0) {
				init(&barrier, group.size());
			}
			group.sync();
			part_index = 0;
			while (part_index < KICK_PP_MAX && i < partsz) {
				while (i + 1 < partsz) {
					const auto other_tree_parts = tree_nodes[partlist[i + 1]].part_range;
					if (these_parts.second == other_tree_parts.first) {
						these_parts.second = other_tree_parts.second;
						i++;
					} else {
						break;
					}
				}
				const part_int imin = these_parts.first;
				const part_int imax = min(these_parts.first + (KICK_PP_MAX - part_index), these_parts.second);
				const int sz = imax - imin;
				cuda::memcpy_async(group, src_x.data() + part_index, main_src_x + imin, sizeof(fixed32) * sz, barrier);
				cuda::memcpy_async(group, src_y.data() + part_index, main_src_y + imin, sizeof(fixed32) * sz, barrier);
				cuda::memcpy_async(group, src_z.data() + part_index, main_src_z + imin, sizeof(fixed32) * sz, barrier);
				__syncwarp();
				these_parts.first += sz;
				part_index += sz;
				if (these_parts.first == these_parts.second) {
					i++;
					if (i < partsz) {
						these_parts = tree_nodes[partlist[i]].part_range;
					}
				}
			}
			barrier.arrive_and_wait();
			__syncwarp();
			for (int j = tid; j < part_index; j += warpSize) {
				sfmm::vec3<float> dx;
				dx[XDIM] = distance(self.mpos->pos[XDIM], src_x[j]);
				dx[YDIM] = distance(self.mpos->pos[YDIM], src_y[j]);
				dx[ZDIM] = distance(self.mpos->pos[ZDIM], src_z[j]);
				flops += P2L_ewald(L, 1.0, dx, do_phi);
			}
		}
		shared_reduce_add_array<float, EXPANSION_SIZE>(L.data());
		for (int i = tid; i < EXPANSION_SIZE; i += WARP_SIZE) {
			Lacc[i] += L[i];
		}
		flops += 6 * EXPANSION_SIZE;

		__syncwarp();
	}
	add_gpu_flops(flops);
}

__device__
void cuda_gravity_pc_ewald(const cuda_kick_data& data, const tree_node& self, const device_vector<int>& multlist, bool do_phi) {
	const int &tid = threadIdx.x;
	__shared__
	extern int shmem_ptr[];
	cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
	auto &force = shmem.f;
	auto& multis = shmem.mpos;
	const int nsink = self.part_range.second - self.part_range.first;
	const auto* sink_x = data.x + self.part_range.first;
	const auto* sink_y = data.y + self.part_range.first;
	const auto* sink_z = data.z + self.part_range.first;
	auto& barrier = shmem.barrier;
	const auto* tree_nodes = data.tree_nodes;
	flop_counter<int> flops = 0;
	auto group = cooperative_groups::this_thread_block();
	if (multlist.size()) {
		__syncwarp();
		int mi = 0;
		const int cnt = multlist.size();
		while (mi < cnt) {
			int mend = min(cnt, mi + KICK_C_MAX);
			for (int this_mi = mi; this_mi < mend; this_mi++) {
				cuda::memcpy_async(group, (void*) &multis[this_mi - mi], tree_nodes[multlist[this_mi]].mpos, sizeof(multi_pos), barrier);
			}
			mend -= mi;
			mi += mend;
			barrier.arrive_and_wait();
			force_type<float> f;
			sfmm::vec3<float> dx;

			for (int j = 0; j < mend; j++) {
				const auto& pos = multis[j].pos;
				const auto& M = multis[j].multi;
				for (int k = tid; k < nsink; k += WARP_SIZE) {
					auto& F = force[k];
					f.init();
					dx[XDIM] = distance(sink_x[k], pos[XDIM]);
					dx[YDIM] = distance(sink_y[k], pos[YDIM]);
					dx[ZDIM] = distance(sink_z[k], pos[ZDIM]);
					flops += M2P(f, M, dx, do_phi);
					F += f;
					flops += 4;
				}
			}
			__syncwarp();
		}
	}
	__syncwarp();
	add_gpu_flops(flops);
}

__device__
void cuda_gravity_pp_ewald(const cuda_kick_data& data, const tree_node& self, const device_vector<int>& partlist, float h, bool do_phi) {
	const int &tid = threadIdx.x;
	__shared__
	extern int shmem_ptr[];
	cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
	auto &force = shmem.f;
	const int nsink = self.part_range.second - self.part_range.first;
	const auto* sink_x = data.x + self.part_range.first;
	const auto* sink_y = data.y + self.part_range.first;
	const auto* sink_z = data.z + self.part_range.first;
	auto& barrier = shmem.barrier;
	const auto* main_src_x = data.x;
	const auto* main_src_y = data.y;
	const auto* main_src_z = data.z;
	auto& src_x = shmem.X.x;
	auto& src_y = shmem.X.y;
	auto& src_z = shmem.X.z;
	const auto* tree_nodes = data.tree_nodes;
	int part_index;
	const float h2 = sqr(h);
	const float hinv = 1.f / h;
	const float h2inv = sqr(hinv);
	const float h3inv = h2inv * hinv;
	flop_counter<int> flops = 7;
	int close = 0;
	int direct = 0;
	if (partlist.size()) {
		int i = 0;
		auto these_parts = tree_nodes[partlist[0]].part_range;
		const auto partsz = partlist.size();
		while (i < partsz) {
			auto group = cooperative_groups::this_thread_block();
			part_index = 0;
			while (part_index < KICK_PP_MAX && i < partsz) {
				while (i + 1 < partsz) {
					const auto other_tree_parts = tree_nodes[partlist[i + 1]].part_range;
					if (these_parts.second == other_tree_parts.first) {
						these_parts.second = other_tree_parts.second;
						i++;
					} else {
						break;
					}
				}
				const part_int imin = these_parts.first;
				const part_int imax = min(these_parts.first + (KICK_PP_MAX - part_index), these_parts.second);
				const int sz = imax - imin;
				cuda::memcpy_async(group, src_x.data() + part_index, main_src_x + imin, sizeof(fixed32) * sz, barrier);
				cuda::memcpy_async(group, src_y.data() + part_index, main_src_y + imin, sizeof(fixed32) * sz, barrier);
				cuda::memcpy_async(group, src_z.data() + part_index, main_src_z + imin, sizeof(fixed32) * sz, barrier);
				__syncwarp();
				these_parts.first += sz;
				part_index += sz;
				if (these_parts.first == these_parts.second) {
					i++;
					if (i < partsz) {
						these_parts = tree_nodes[partlist[i]].part_range;
					}
				}
			}
			barrier.arrive_and_wait();
			__syncwarp();
			force_type<float> f;
			sfmm::vec3<float> dx;
			for (int k = tid; k < nsink; k += WARP_SIZE) {
				f.init();
				for (int j = 0; j < part_index; j++) {
					dx[0] = distance(sink_x[k], src_x[j]); // 1
					dx[1] = distance(sink_y[k], src_y[j]); // 1
					dx[2] = distance(sink_z[k], src_z[j]); // 1
					flops += P2P_ewald(f, 1.f, dx);
				}
				auto& F = force[k];
				F += f;
			}
		}
		__syncwarp();
	}
	__syncwarp();
	add_gpu_flops(flops);
}
#endif
