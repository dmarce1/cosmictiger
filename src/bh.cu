#include "hip/hip_runtime.h"
#include <cosmictiger/bh.hpp>
#include <cosmictiger/cuda.hpp>
#include <cosmictiger/cuda_reduce.hpp>
#include <cosmictiger/hpx.hpp>
#include <cosmictiger/gravity.hpp>
#include <cosmictiger/options.hpp>
#include <cosmictiger/math.hpp>

#define BH_LIST_SIZE 32767

struct bh_lists {
	fixedcapvec<int, 4095> checklist;
	fixedcapvec<int, 2048> nextlist;
	fixedcapvec<bh_source, 8192> sourcelist;
	fixedcapvec<int, 4096> leaflist;
};

__global__ void bh_kernel(bh_lists* lists, bh_tree_node* nodes, int* sink_buckets, array<float, NDIM>* parts, float* phi, int nsink_buckets, int* current,
		float theta, float h, float GM) {

	const int& tid = threadIdx.x;
	const int& bid = blockIdx.x;
	__shared__ int index;
	const float thetainv = 1.f / theta;
	const float hinv = 1.0f / h;
	const float h2 = h * h;
	const float h2inv = hinv * hinv;

	if (tid == 0) {
		index = atomicAdd(current, 1);
	}
	__syncwarp();
	while (index < nsink_buckets) {
		auto& checklist = lists[bid].checklist;
		auto& nextlist = lists[bid].nextlist;
		auto& sourcelist = lists[bid].sourcelist;
		auto& leaflist = lists[bid].leaflist;
		nextlist.resize(0);
		sourcelist.resize(0);
		leaflist.resize(0);
		checklist.resize(1);
		checklist[0] = 0;
		const auto& mynode = nodes[sink_buckets[index]];
		const float& myradius = mynode.radius;
		const auto& mypos = mynode.pos;
		const auto& myparts = mynode.parts;
		for (int i = myparts.first + tid; i < myparts.second; i += WARP_SIZE) {
			phi[i] = -SELF_PHI * hinv;
		}
		while (checklist.size()) {
			const int maxi = round_up(checklist.size(), WARP_SIZE);
			for (int ci = tid; ci < maxi; ci += WARP_SIZE) {
				bool next = false;
				bool pc = false;
				bool pp = false;
				float r2;
				const bh_tree_node* node_ptr;
				if (ci < checklist.size()) {
					node_ptr = &nodes[checklist[ci]];
					const float dx = mypos[XDIM] - node_ptr->pos[XDIM];
					const float dy = mypos[YDIM] - node_ptr->pos[YDIM];
					const float dz = mypos[ZDIM] - node_ptr->pos[ZDIM];
					r2 = sqr(dx, dy, dz);
					if (r2 > sqr(thetainv * (node_ptr->radius + myradius))) {
						pc = true;
					} else if (node_ptr->children[LEFT] == -1) {
						pp = true;
					} else {
						next = true;
					}
				}
				int total;
				int index;

				index = next;
				compute_indices(index, total);
				index += nextlist.size();
				nextlist.resize(nextlist.size() + total);
				if (next) {
					nextlist[index] = checklist[ci];
				}

				index = pc;
				compute_indices(index, total);
				index += sourcelist.size();
				sourcelist.resize(sourcelist.size() + total);
				if (pc) {
					bh_source src;
					src.x = node_ptr->pos;
					src.m = node_ptr->mass;
					sourcelist[index] = src;
				}

				index = pp;
				compute_indices(index, total);
				index += leaflist.size();
				leaflist.resize(leaflist.size() + total);
				if (pp) {
					leaflist[index] = checklist[ci];
				}

			}
			checklist.resize(NCHILD * nextlist.size());
			for (int i = tid; i < nextlist.size(); i += WARP_SIZE) {
				const auto& this_node = nodes[nextlist[i]];
				checklist[NCHILD * i + LEFT] = this_node.children[LEFT];
				checklist[NCHILD * i + RIGHT] = this_node.children[RIGHT];
			}
			nextlist.resize(0);
			for (int i = 0; i < leaflist.size(); i++) {
				const auto& other_node = nodes[leaflist[i]];
				const auto& other_parts = other_node.parts;
				const auto other_size = other_parts.second - other_parts.first;
				const int offset = sourcelist.size();
				sourcelist.resize(other_size + offset);
				for (int j = other_parts.first + tid; i < other_parts.second; i += WARP_SIZE) {
					bh_source src;
					src.x = parts[j];
					src.m = 1.f;
					sourcelist[offset + j - other_parts.first] = src;
				}
			}
			leaflist.resize(0);
			__syncwarp();

		}
		for (int i = 0; i < sourcelist.size(); i++) {
			for (int j = myparts.first + tid; j < myparts.second; j += WARP_SIZE) {
				const auto& sink = parts[j];
				const auto& source = sourcelist[i];
				const auto& m = source.m;
				const float dx = sink[XDIM] - source.x[XDIM];
				const float dy = sink[YDIM] - source.x[YDIM];
				const float dz = sink[ZDIM] - source.x[ZDIM];
				const float r2 = sqr(dx, dy, dz);
				if (r2 > h2) {
					phi[j] -= m * rsqrt(r2);
				} else {
					const float q2 = r2 * h2inv;
					float rinv = -5.0f / 16.0f;
					rinv = fmaf(rinv, q2, 21.0f / 16.0f);
					rinv = fmaf(rinv, q2, -35.0f / 16.0f);
					rinv = fmaf(rinv, q2, 35.0f / 16.0f);
					rinv *= hinv;
					phi[j] -= m * rinv;
				}

			}
		}
		for (int i = myparts.first + tid; i < myparts.second; i += WARP_SIZE) {
			phi[i] *= GM;
		}

		if (tid == 0) {
			index = atomicAdd(current, 1);
		}
		__syncwarp();
	}

}

vector<float> bh_cuda_tree_evaluate(const vector<bh_tree_node>& nodes, vector<int>& sink_buckets, vector<array<float, NDIM>>& parts, float theta) {
	vector<float> phi;
//	PRINT( "%i %i\n", sinks.size(), nodes.size());
	bh_tree_node* dev_nodes;
	array<float, NDIM>* dev_parts;
	float* dev_phi;
	int* dev_current;
	int* dev_sink_buckets;
	int zero = 0;
	bh_lists* dev_lists;
	int blocks_per;
	int max_blocks;
	CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blocks_per, (const void*) bh_kernel, WARP_SIZE, sizeof(int)));
	max_blocks = cuda_smp_count() * blocks_per;
//	blocks *= std::min(std::max((cuda_smp_count() - 1) / hpx_hardware_concurrency() + 1, 1), max_blocks);
	int blocks = std::min((int) sink_buckets.size(), max_blocks);
	CUDA_CHECK(hipMalloc(&dev_lists, sizeof(bh_lists) * blocks));
	CUDA_CHECK(hipMalloc(&dev_nodes, sizeof(bh_tree_node) * nodes.size()));
	CUDA_CHECK(hipMalloc(&dev_phi, sizeof(float) * parts.size()));
	CUDA_CHECK(hipMalloc(&dev_current, sizeof(int)));
	CUDA_CHECK(hipMalloc(&dev_sink_buckets, sizeof(int) * sink_buckets.size()));
	CUDA_CHECK(hipMalloc(&dev_parts, sizeof(array<float, NDIM> ) * parts.size()));
	auto stream = cuda_get_stream();
	CUDA_CHECK(hipMemcpyAsync(dev_nodes, nodes.data(), sizeof(bh_tree_node) * nodes.size(), hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(dev_parts, parts.data(), sizeof(array<float, NDIM> ) * parts.size(), hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(dev_sink_buckets, sink_buckets.data(), sizeof(int) * sink_buckets.size(), hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(dev_current, &zero, sizeof(int), hipMemcpyHostToDevice, stream));
	bh_kernel<<<blocks,WARP_SIZE,0,stream>>>(dev_lists, dev_nodes, dev_sink_buckets, dev_parts, dev_phi, sink_buckets.size(), dev_current, 0.5, get_options().hsoft, get_options().GM);
	phi.resize(parts.size());
	CUDA_CHECK(hipMemcpyAsync(phi.data(), dev_phi, sizeof(float) * phi.size(), hipMemcpyDeviceToHost, stream));
	while (hipStreamQuery(stream) != hipSuccess) {
		hpx_yield();
	}
	cuda_end_stream(stream);
	CUDA_CHECK(hipFree(dev_nodes));
	CUDA_CHECK(hipFree(dev_parts));
	CUDA_CHECK(hipFree(dev_current));
	CUDA_CHECK(hipFree(dev_lists));

	return phi;
}

