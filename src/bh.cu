#include "hip/hip_runtime.h"
#include <cosmictiger/bh.hpp>
#include <cosmictiger/cuda.hpp>
#include <cosmictiger/cuda_reduce.hpp>
#include <cosmictiger/hpx.hpp>
#include <cosmictiger/gravity.hpp>
#include <cosmictiger/options.hpp>
#include <cosmictiger/math.hpp>

#define BH_LIST_SIZE 32767

struct bh_lists {
	fixedcapvec<int, 2048> checklist;
	fixedcapvec<int, 1024> nextlist;
	fixedcapvec<bh_source, 16384> sourcelist;
};

__global__ void bh_kernel(bh_lists* lists, bh_tree_node* nodes, int* sink_buckets, array<float, NDIM>* parts, float* phi, int nsink_buckets, int* current,
		float theta, float h, float GM) {

	const int& tid = threadIdx.x;
	const int& bid = blockIdx.x;
	__shared__ int index;
	const float thetainv = 1.f / theta;
	const float hinv = 1.0f / h;
	const float h2 = h * h;
	const float h2inv = hinv * hinv;

	if (tid == 0) {
		index = atomicAdd(current, 1);
	}
	__syncwarp();
	while (index < nsink_buckets) {
		auto& checklist = lists[bid].checklist;
		auto& nextlist = lists[bid].nextlist;
		auto& sourcelist = lists[bid].sourcelist;
		nextlist.resize(0);
		sourcelist.resize(0);
		checklist.resize(1);
		checklist[0] = 0;
		const auto& mynode = nodes[sink_buckets[index]];
		const float& myradius = mynode.radius;
		const auto& mypos = mynode.pos;
		const auto& myparts = mynode.parts;
		for (int i = myparts.first + tid; i < myparts.second; i += WARP_SIZE) {
			phi[i] = -SELF_PHI * hinv;
		}
		while (checklist.size()) {
			const int maxi = round_up(checklist.size(), WARP_SIZE);
			for (int ci = tid; ci < maxi; ci += WARP_SIZE) {
				bool next = false;
				bool pc = false;
				bool pp = false;
				float r2;
				const bh_tree_node* node_ptr;
				if (ci < checklist.size()) {
					node_ptr = &nodes[checklist[ci]];
					const float dx = mypos[XDIM] - node_ptr->pos[XDIM];
					const float dy = mypos[YDIM] - node_ptr->pos[YDIM];
					const float dz = mypos[ZDIM] - node_ptr->pos[ZDIM];
					r2 = sqr(dx, dy, dz);
					if (r2 > thetainv * (node_ptr->radius + mynode.radius)) {
						pc = true;
					} else if (node_ptr->children[LEFT] == -1) {
						pp = true;
					} else {
						next = true;
					}
				}
				int total;
				int index;

				index = next;
				compute_indices(index, total);
				index += nextlist.size();
				nextlist.resize(nextlist.size() + total);
				if (next) {
					nextlist[index] = checklist[ci];
				}

				index = pc;
				compute_indices(index, total);
				index += sourcelist.size();
				sourcelist.resize(sourcelist.size() + total);
				if (pc) {
					bh_source src;
					src.x = node_ptr->pos;
					src.m = node_ptr->mass;
				}

			}
			checklist.resize(NCHILD * nextlist.size());
			for (int i = tid; i < nextlist.size(); i += WARP_SIZE) {
				const auto& this_node = nodes[nextlist[i]];
				checklist[NCHILD * i + LEFT] = this_node.children[LEFT];
				checklist[NCHILD * i + RIGHT] = this_node.children[RIGHT];
			}

		}

		/*	 const array<float, NDIM>& sink = sinks[index];
		 phi[index] = -SELF_PHI * hinv;
		 int depth = 0;
		 while (checklist.size()) {
		 const int maxi = round_up(checklist.size(), WARP_SIZE);
		 for (int ci = tid; ci < maxi; ci += WARP_SIZE) {
		 bool next = false;
		 bool interact = false;
		 float r2;
		 const bh_tree_node* node_ptr;
		 if (ci < checklist.size()) {
		 node_ptr = &nodes[checklist[ci]];
		 if (node_ptr->count) {
		 const float dx = sink[XDIM] - node_ptr->pos[XDIM];
		 const float dy = sink[YDIM] - node_ptr->pos[YDIM];
		 const float dz = sink[ZDIM] - node_ptr->pos[ZDIM];
		 r2 = sqr(dx, dy, dz);
		 if ((node_ptr->children[LEFT] == -1) || (r2 > thetainv * node_ptr->radius)) {
		 interact = true;
		 } else {
		 next = true;
		 }
		 }
		 }
		 int total;
		 int index;

		 index = next;
		 compute_indices(index, total);
		 index += nextlist.size();
		 nextlist.resize(nextlist.size() + total);
		 if (next) {
		 nextlist[index] = checklist[ci];
		 }

		 index = interact;
		 compute_indices(index, total);
		 index += dist2list.size();
		 dist2list.resize(dist2list.size() + total);
		 masslist.resize(masslist.size() + total);
		 if (interact) {
		 dist2list[index] = r2;
		 masslist[index] = node_ptr->count;
		 }

		 }
		 checklist.resize(NCHILD * nextlist.size());
		 for (int i = tid; i < nextlist.size(); i += WARP_SIZE) {
		 const auto& this_node = nodes[nextlist[i]];
		 checklist[NCHILD * i + LEFT] = this_node.children[LEFT];
		 checklist[NCHILD * i + RIGHT] = this_node.children[RIGHT];
		 }
		 float this_phi = 0.f;
		 for (int ci = tid; ci < dist2list.size(); ci += WARP_SIZE) {
		 const auto& r2 = dist2list[ci];
		 const float m = masslist[ci];
		 if (r2 > h2) {
		 this_phi -= m * rsqrtf(dist2list[ci]);
		 } else {
		 const float q2 = r2 * h2inv;
		 float rinv = -5.0f / 16.0f;
		 rinv = fmaf(rinv, q2, 21.0f / 16.0f);
		 rinv = fmaf(rinv, q2, -35.0f / 16.0f);
		 rinv = fmaf(rinv, q2, 35.0f / 16.0f);
		 rinv *= hinv;
		 this_phi -= m * rinv;
		 }
		 }
		 shared_reduce_add(this_phi);
		 if (tid == 0) {
		 phi[index] += this_phi;
		 }
		 __syncwarp();
		 dist2list.resize(0);
		 masslist.resize(0);
		 nextlist.resize(0);
		 depth++;
		 }
		 phi[index] *= GM;*/
		if (tid == 0) {
			index = atomicAdd(current, 1);
		}
		__syncwarp();
	}

}

vector<float> bh_cuda_tree_evaluate(const vector<bh_tree_node>& nodes, vector<int>& sink_buckets, vector<array<float, NDIM>>& parts, float theta) {
	vector<float> phi;
//	PRINT( "%i %i\n", sinks.size(), nodes.size());
	bh_tree_node* dev_nodes;
	array<float, NDIM>* dev_parts;
	float* dev_phi;
	int* dev_current;
	int* dev_sink_buckets;
	int zero = 0;
	bh_lists* dev_lists;
	int blocks_per;
	int max_blocks;
	CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blocks_per, (const void*) bh_kernel, WARP_SIZE, sizeof(int)));
	max_blocks = cuda_smp_count() * blocks_per;
//	blocks *= std::min(std::max((cuda_smp_count() - 1) / hpx_hardware_concurrency() + 1, 1), max_blocks);
	int blocks = std::min((int) (((sink_buckets.size() - 1) / blocks_per + 1) * blocks_per), max_blocks);
	CUDA_CHECK(hipMalloc(&dev_lists, sizeof(bh_lists) * blocks));
	CUDA_CHECK(hipMalloc(&dev_nodes, sizeof(bh_tree_node) * nodes.size()));
	CUDA_CHECK(hipMalloc(&dev_phi, sizeof(float) * parts.size()));
	CUDA_CHECK(hipMalloc(&dev_current, sizeof(int)));
	CUDA_CHECK(hipMalloc(&dev_sink_buckets, sizeof(int) * sink_buckets.size()));
	CUDA_CHECK(hipMalloc(&dev_parts, sizeof(array<float, NDIM> ) * parts.size()));
	auto stream = cuda_get_stream();
	CUDA_CHECK(hipMemcpyAsync(dev_nodes, nodes.data(), sizeof(bh_tree_node) * nodes.size(), hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(dev_parts, parts.data(), sizeof(array<float, NDIM> ) * parts.size(), hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(dev_sink_buckets, sink_buckets.data(), sizeof(int) * sink_buckets.size(), hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(dev_current, &zero, sizeof(int), hipMemcpyHostToDevice, stream));
	bh_kernel<<<blocks,WARP_SIZE,0,stream>>>(dev_lists, dev_nodes, dev_sink_buckets, dev_parts, dev_phi, sink_buckets.size(), dev_current, 0.5, get_options().hsoft, get_options().GM);
	phi.resize(parts.size());
	CUDA_CHECK(hipMemcpyAsync(phi.data(), dev_phi, sizeof(float) * phi.size(), hipMemcpyDeviceToHost, stream));
	while (hipStreamQuery(stream) != hipSuccess) {
		hpx_yield();
	}
	cuda_end_stream(stream);
	CUDA_CHECK(hipFree(dev_nodes));
	CUDA_CHECK(hipFree(dev_parts));
	CUDA_CHECK(hipFree(dev_current));
	CUDA_CHECK(hipFree(dev_lists));

	return phi;
}

