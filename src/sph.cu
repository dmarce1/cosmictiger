#include "hip/hip_runtime.h"
/*
 CosmicTiger - A cosmological N-Body code
 Copyright (C) 2021  Dominic C. Marcello

 This program is free software; you can redistribute it and/or
 modify it under the terms of the GNU General Public License
 as published by the Free Software Foundation; either version 2
 of the License, or (at your option) any later version.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program; if not, write to the Free Software
 Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 */

struct smoothlen_shmem {
	int index;
};

#include <cosmictiger/sph_cuda.hpp>
#include <cosmictiger/cuda_reduce.hpp>

#define WORKSPACE_SIZE (256*SPH_NEIGHBOR_COUNT)

struct smoothlen_workspace {
	fixedcapvec<fixed32, WORKSPACE_SIZE> x;
	fixedcapvec<fixed32, WORKSPACE_SIZE> y;
	fixedcapvec<fixed32, WORKSPACE_SIZE> z;
};

__global__ void sph_cuda_smoothlen(sph_run_params params, sph_run_cuda_data data, smoothlen_workspace* workspaces, int* counter, int* flag) {
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int block_size = blockDim.x;
	__shared__
	extern int shmem_ptr[];
	smoothlen_shmem &shmem = *(smoothlen_shmem*) shmem_ptr;
	smoothlen_workspace& ws = workspaces[bid];
	auto& index = shmem.index;
	if (tid == 0) {
		index = atomicAdd(counter, 1);
	}
	__syncwarp();
	array<fixed32, NDIM> x;
	while (index < data.nselfs) {
		ws.x.resize(0);
		ws.y.resize(0);
		ws.z.resize(0);
		const sph_tree_node& self = data.trees[data.selfs[index]];
	//	PRINT( "%i\n", self.neighbor_range.second - self.neighbor_range.first);
		for (int ni = self.neighbor_range.first; ni < self.neighbor_range.second; ni++) {
			//PRINT( "%i\n", -self.neighbor_range.first+self.neighbor_range.second);
			const sph_tree_node& other = data.trees[data.neighbors[ni]];
			const int maxpi = round_up(other.part_range.second - other.part_range.first, block_size) + other.part_range.first;
			for (int pi = other.part_range.first + tid; pi < maxpi; pi += block_size) {
				bool contains = false;
				int j;
				int total;
				if (pi < other.part_range.second) {
					x[XDIM] = data.x[pi];
					x[YDIM] = data.y[pi];
					x[ZDIM] = data.z[pi];
				//	contains = true;
					if (self.outer_box.contains(x)) {
						contains = true;
			//			if( tid == 0 ) {
			//				PRINT( "TRUE\n");
			//			}
					} /*else if( tid == 0 ) {
						PRINT( "FALSE\n");
					}*/
				}
				j = contains;
				compute_indices(j, total);
				const int offset = ws.x.size();
				const int next_size = offset + total;
				ws.x.resize(next_size);
				ws.y.resize(next_size);
				ws.z.resize(next_size);
				if (contains) {
					const int k = offset + j;
					ws.x[k] = x[XDIM];
					ws.y[k] = x[YDIM];
					ws.z[k] = x[ZDIM];
				}
			}
		}

		constexpr float A = float(float(21.0 * 2.0 / 3.0));
		constexpr float B = float(float(840.0 / 3.0));
		if( ws.x.size() == 0 ) {
			PRINT( "ZERO\n");
		}
		for (int i = self.part_range.first; i < self.part_range.second; i++) {
			if (data.rungs[i] >= params.min_rung) {
				const int snki = self.sink_part_range.first - self.part_range.first + i;
				x[XDIM] = data.x[i];
				x[YDIM] = data.y[i];
				x[ZDIM] = data.z[i];
				float error;
				int count;
				float f;
				float dfdh;
				int box_xceeded = false;
				int iter = 0;
				float dh;
				float& h = data.h_snk[snki];
				do {
					const float hinv = 1.f / h;
					const float h2 = sqr(h);
					count = 0;
					f = 0.f;
					dfdh = 0.f;
					if( ws.x.size() == 0 ) {
				//		PRINT( "ZERO\n");
					}
					for (int j = tid; j < ws.x.size(); j += block_size) {
						const float dx = distance(x[XDIM], ws.x[j]);
						const float dy = distance(x[YDIM], ws.y[j]);
						const float dz = distance(x[ZDIM], ws.z[j]);
						const float r2 = sqr(dx, dy, dz);
						count += int(r2 < h2);
						const float r = sqrt(r2);
						const float q = r * hinv;
						if (q < 1.f) {
							const float q2 = sqr(q);
							const float _1mq = 1.f - q;
							const float _1mq2 = sqr(_1mq);
							const float _1mq3 = _1mq * _1mq2;
							const float _1mq4 = _1mq * _1mq3;
							const float w = A * _1mq4 * fmaf(4.f, q, 1.f);
							const float dwdh = B * _1mq3 * q2 * hinv;
							f += w;
							dfdh += dwdh;
						}
					}
					shared_reduce_add(f);
					shared_reduce_add(count);
					shared_reduce_add(dfdh);
					dh = 0.1f * h;
					if (count > 1) {
						f -= SPH_NEIGHBOR_COUNT;
						dh = -f / dfdh;
						if (dh > 0.5f * h) {
							dh = 0.5f * h;
						} else if (dh < -0.5f * h) {
							dh = -0.5f * h;
						}
						error = fabsf(logf(h + dh) - logf(h));
						if (tid == 0) {
							h += dh;
						}
					} else {
						if( count == 0 ) {
							PRINT( "Can't find self\n");
						}
						if (tid == 0) {
							h *= 1.1;
						}
						error = 1.0;
					}
					__syncwarp();
					for (int dim = 0; dim < NDIM; dim++) {
						if (distance(self.outer_box.end[dim], x[dim]) + h < 0.0f) {
							box_xceeded = true;
							break;
						}
						if (distance(x[dim], self.outer_box.begin[dim]) + h < 0.0f) {
							box_xceeded = true;
							break;
						}
					}
					iter++;
		//			if( tid == 0 )
		//			PRINT("%i %i %e %e\n", iter, count, h, dh);
					if (iter > 20) {
					//	PRINT("density solver failed to converge\n");
					//	__trap();
					}
				} while (error > SPH_SMOOTHLEN_TOLER && !box_xceeded);
				if (tid == 0 && box_xceeded) {
					atomicAdd(flag, 1);
				}
			}
		}

		if (tid == 0) {
			index = atomicAdd(counter, 1);
		}
		__syncwarp();
	}

}

sph_run_return sph_run_cuda(sph_run_params params, sph_run_cuda_data data, hipStream_t stream) {
	sph_run_return rc;
	int* counter;
	int* flag;
	int nblocks;
	CUDA_CHECK(hipMallocManaged(&counter, sizeof(int)));
	CUDA_CHECK(hipMallocManaged(&flag, sizeof(int)));
	*counter = 0;
	*flag = 0;
	switch (params.run_type) {
	case SPH_RUN_SMOOTHLEN: {
		smoothlen_workspace* workspaces;
		CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&nblocks, (const void*) sph_cuda_smoothlen, WARP_SIZE, sizeof(smoothlen_shmem)));
		nblocks *= cuda_smp_count();
		CUDA_CHECK(hipMalloc(&workspaces, sizeof(smoothlen_workspace) * nblocks));
		sph_cuda_smoothlen<<<nblocks, WARP_SIZE>>>(params,data,workspaces,counter, flag);
		cuda_stream_synchronize(stream);
		CUDA_CHECK(hipFree(workspaces));
		rc.rc = flag;
	}
		break;
	}
	CUDA_CHECK(hipFree(counter));
	CUDA_CHECK(hipFree(flag));

	return rc;
}
