#include "hip/hip_runtime.h"
/*
 CosmicTiger - A cosmological N-Body code
 Copyright (C) 2021  Dominic C. Marcello

 This program is free software; you can redistribute it and/or
 modify it under the terms of the GNU General Public License
 as published by the Free Software Foundation; either version 2
 of the License, or (at your option) any later version.

 This program is distribufted in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program; if not, write to the Free Software
 Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 */

#define SPH_DIFFUSION_C 0.03f
#define SMOOTHLEN_BLOCK_SIZE 256
#define MARK_SEMIACTIVE_BLOCK_SIZE 256
#define RUNGS_BLOCK_SIZE 256
#define XSPH_BLOCK_SIZE 256
#define HYDRO_BLOCK_SIZE 128
#define AUX_BLOCK_SIZE 128

#define SPH_SMOOTHLEN_TOLER float(1.0e-5)

struct smoothlen_shmem {
	int index;
};

#include <cosmictiger/sph_cuda.hpp>
#include <cosmictiger/cuda_mem.hpp>
#include <cosmictiger/cuda_reduce.hpp>
#include <cosmictiger/constants.hpp>
#include <cosmictiger/timer.hpp>
#include <cosmictiger/kernel.hpp>

#include <cosmictiger/math.hpp>

static __constant__ float rung_dt[MAX_RUNG] = { 1.0 / (1 << 0), 1.0 / (1 << 1), 1.0 / (1 << 2), 1.0 / (1 << 3), 1.0 / (1 << 4), 1.0 / (1 << 5), 1.0 / (1 << 6),
		1.0 / (1 << 7), 1.0 / (1 << 8), 1.0 / (1 << 9), 1.0 / (1 << 10), 1.0 / (1 << 11), 1.0 / (1 << 12), 1.0 / (1 << 13), 1.0 / (1 << 14), 1.0 / (1 << 15), 1.0
				/ (1 << 16), 1.0 / (1 << 17), 1.0 / (1 << 18), 1.0 / (1 << 19), 1.0 / (1 << 20), 1.0 / (1 << 21), 1.0 / (1 << 22), 1.0 / (1 << 23), 1.0 / (1 << 24),
		1.0 / (1 << 25), 1.0 / (1 << 26), 1.0 / (1 << 27), 1.0 / (1 << 28), 1.0 / (1 << 29), 1.0 / (1 << 30), 1.0 / (1 << 31) };

struct smoothlen_workspace {
	device_vector<fixed32> x;
	device_vector<fixed32> y;
	device_vector<fixed32> z;
};

struct mark_semiactive_workspace {
	device_vector<fixed32> x;
	device_vector<fixed32> y;
	device_vector<fixed32> z;
	device_vector<float> h;
	device_vector<char> rungs;
};

struct rungs_workspace {
	device_vector<fixed32> x;
	device_vector<fixed32> y;
	device_vector<fixed32> z;
	device_vector<float> h;
	device_vector<char> rungs;
};

struct xsph_record1 {
	fixed32 x;
	fixed32 y;
	fixed32 z;
};

struct xsph_record2 {
	float vx;
	float vy;
	float vz;
};

struct hydro_record1 {
	fixed32 x;
	fixed32 y;
	fixed32 z;
	float h;
};

struct hydro_record2 {
	array<float, NCHEMFRACS> chem;
	float shearv;
	float vx;
	float vy;
	float vz;
	float entr;
	float alpha;
	float fpre;
	float cold_frac;
	char rung;
};

struct aux_record1 {
	fixed32 x;
	fixed32 y;
	fixed32 z;
};

struct aux_record2 {
	float vx;
	float vy;
	float vz;
	float entr;
	float gamma;
	float h;
};

struct hydro_workspace {
	device_vector<hydro_record1> rec1;
	device_vector<hydro_record2> rec2;
};

struct xsph_workspace {
	device_vector<xsph_record1> rec1;
	device_vector<xsph_record2> rec2;
};

struct aux_workspace {
	device_vector<aux_record1> rec1;
	device_vector<aux_record2> rec2;
};

struct sph_reduction {
	int counter;
	int flag;
	float hmin;
	float hmax;
	float vsig_max;
	float flops;
	int max_rung_hydro;
	int max_rung_grav;
	int max_rung;
};

__global__ void sph_cuda_xsph(sph_run_params params, sph_run_cuda_data data, sph_reduction* reduce) {
	const int tid = threadIdx.x;
	const int block_size = blockDim.x;
	__shared__
	int index;
	__shared__ xsph_workspace ws;
	__syncthreads();
	if (tid == 0) {
		index = atomicAdd(&reduce->counter, 1);
	}
	__syncthreads();
	new (&ws) xsph_workspace();
	array<fixed32, NDIM> x;
	while (index < data.nselfs) {

		ws.rec1.resize(0);
		ws.rec2.resize(0);
		const sph_tree_node& self = data.trees[data.selfs[index]];
		for (int ni = self.neighbor_range.first; ni < self.neighbor_range.second; ni++) {
			const sph_tree_node& other = data.trees[data.neighbors[ni]];
			const int maxpi = round_up(other.part_range.second - other.part_range.first, block_size) + other.part_range.first;
			for (int pi = other.part_range.first + tid; pi < maxpi; pi += block_size) {
				bool contains = false;
				int j;
				int total;
				if (pi < other.part_range.second) {
					x[XDIM] = data.x[pi];
					x[YDIM] = data.y[pi];
					x[ZDIM] = data.z[pi];
					contains = (self.outer_box.contains(x));
				}
				j = contains;
				compute_indices<XSPH_BLOCK_SIZE>(j, total);
				const int offset = ws.rec1.size();
				__syncthreads();
				const int next_size = offset + total;
				ws.rec1.resize(next_size);
				ws.rec2.resize(next_size);
				if (contains) {
					const int k = offset + j;
					ws.rec1[k].x = x[XDIM];
					ws.rec1[k].y = x[YDIM];
					ws.rec1[k].z = x[ZDIM];
					ws.rec2[k].vx = data.vx[pi];
					ws.rec2[k].vy = data.vy[pi];
					ws.rec2[k].vz = data.vz[pi];
				}
			}
		}
		for (int i = self.part_range.first; i < self.part_range.second; i++) {
			__syncthreads();
			const int snki = self.sink_part_range.first - self.part_range.first + i;
			const bool active = data.rungs[i] >= params.min_rung;
			const bool use = active;
			if (use) {
				const auto x_i = data.x[i];
				const auto y_i = data.y[i];
				const auto z_i = data.z[i];
				const float h_i = data.h_snk[snki];
				const float h2_i = sqr(h_i);
				const auto vx_i = data.vx[i];
				const auto vy_i = data.vy[i];
				const auto vz_i = data.vz[i];
				const float hinv_i = 1.f / h_i;
				float xvx = 0.0f;
				float xvy = 0.0f;
				float xvz = 0.0f;
				for (int j = tid; j < ws.rec1.size(); j += block_size) {
					const auto x_j = ws.rec1[j].x;
					const auto y_j = ws.rec1[j].y;
					const auto z_j = ws.rec1[j].z;
					const float x_ij = distance(x_i, x_j); // 2
					const float y_ij = distance(y_i, y_j); // 2
					const float z_ij = distance(z_i, z_j); // 2
					const float r2 = sqr(x_ij, y_ij, z_ij);            // 2
					if (r2 < h2_i) {
						const float r = sqrt(r2);                    // 4
						const auto& vx_j = ws.rec2[j].vx;
						const auto& vy_j = ws.rec2[j].vy;
						const auto& vz_j = ws.rec2[j].vz;
						const float q_i = r * hinv_i;
						const float W_i = kernelW(q_i);
						xvx = fmaf(W_i, vx_j - vx_i, xvx);
						xvy = fmaf(W_i, vy_j - vy_i, xvy);
						xvz = fmaf(W_i, vz_j - vz_i, xvz);
					}
				}
				shared_reduce_add<float, XSPH_BLOCK_SIZE>(xvx);
				shared_reduce_add<float, XSPH_BLOCK_SIZE>(xvy);
				shared_reduce_add<float, XSPH_BLOCK_SIZE>(xvz);
				if (tid == 0) {
					const float c0 = float(4.0f * M_PI / 3.0f) / data.N;
					xvx *= c0;
					xvy *= c0;
					xvz *= c0;
					data.xvx_snk[snki] = xvx;
					data.xvy_snk[snki] = xvy;
					data.xvz_snk[snki] = xvz;
				}
			}
		}
		if (tid == 0) {
			index = atomicAdd(&reduce->counter, 1);
		}
		__syncthreads();
	}
	(&ws)->~xsph_workspace();
}

__global__ void sph_cuda_smoothlen(sph_run_params params, sph_run_cuda_data data, sph_reduction* reduce) {
	const int tid = threadIdx.x;
	const int block_size = blockDim.x;
	__shared__
	int index;
	__shared__ smoothlen_workspace ws;
	__syncthreads();
	if (tid == 0) {
		index = atomicAdd(&reduce->counter, 1);
	}
	__syncthreads();
	new (&ws) smoothlen_workspace();
	array<fixed32, NDIM> x;
	float error;
	while (index < data.nselfs) {

		int flops = 0;
		ws.x.resize(0);
		ws.y.resize(0);
		ws.z.resize(0);
		const sph_tree_node& self = data.trees[data.selfs[index]];
		for (int ni = self.neighbor_range.first; ni < self.neighbor_range.second; ni++) {
			const sph_tree_node& other = data.trees[data.neighbors[ni]];
			const int maxpi = round_up(other.part_range.second - other.part_range.first, block_size) + other.part_range.first;
			for (int pi = other.part_range.first + tid; pi < maxpi; pi += block_size) {
				bool contains = false;
				int j;
				int total;
				if (pi < other.part_range.second) {
					x[XDIM] = data.x[pi];
					x[YDIM] = data.y[pi];
					x[ZDIM] = data.z[pi];
					if (self.outer_box.contains(x)) {
						contains = true;
					}
				}
				j = contains;
				compute_indices<SMOOTHLEN_BLOCK_SIZE>(j, total);
				const int offset = ws.x.size();
				__syncthreads();
				const int next_size = offset + total;
				ws.x.resize(next_size);
				ws.y.resize(next_size);
				ws.z.resize(next_size);
				if (contains) {
					const int k = offset + j;
					ws.x[k] = x[XDIM];
					ws.y[k] = x[YDIM];
					ws.z[k] = x[ZDIM];
				}
			}
		}
		float hmin = 1e+20;
		float hmax = 0.0;
		for (int i = self.part_range.first; i < self.part_range.second; i++) {
			__syncthreads();
			const int snki = self.sink_part_range.first - self.part_range.first + i;
			const bool active = data.rungs[i] >= params.min_rung;
			const bool converged = data.converged_snk[snki];
			const bool use = active && !converged;
			const float w0 = kernelW(0.f);
			if (use) {
				x[XDIM] = data.x[i];
				x[YDIM] = data.y[i];
				x[ZDIM] = data.z[i];
				int box_xceeded = false;
				int iter = 0;
				float& h = data.h_snk[snki];
				float drho_dh;
				float rhoh3;
				float last_dh = 0.0f;
				float w1 = 1.0f;
				do {
					const float hinv = 1.f / h; // 4
					const float h2 = sqr(h);    // 1
					drho_dh = 0.f;
					rhoh3 = 0.f;
					float rhoh30 = (3.0f * data.N) / (4.0f * float(M_PI));
					for (int j = tid; j < ws.x.size(); j += block_size) {
						const float dx = distance(x[XDIM], ws.x[j]); // 2
						const float dy = distance(x[YDIM], ws.y[j]); // 2
						const float dz = distance(x[ZDIM], ws.z[j]); // 2
						const float r2 = sqr(dx, dy, dz);            // 2
						const float r = sqrt(r2);                    // 4
						const float q = r * hinv;                    // 1
						if (q < 1.f) {                               // 1
							const float w = kernelW(q); // 4
							const float dwdq = dkernelW_dq(q);
							const float dwdh = -q * dwdq * hinv; // 3
							drho_dh -= (3.f * w + q * dwdq);
							rhoh3 += w;
						}

					}
					shared_reduce_add<float, SMOOTHLEN_BLOCK_SIZE>(drho_dh);
					shared_reduce_add<float, SMOOTHLEN_BLOCK_SIZE>(rhoh3);
					float dlogh;
					__syncthreads();
					if (rhoh3 <= w0) {
						if (tid == 0) {
							h *= 1.1f;
						}
						iter--;
						error = 1.0;
					} else {
						drho_dh *= 0.33333333333f / rhoh30;
						const float fpre = fminf(fmaxf(1.0f / (1.0f + drho_dh), 0.5f), 2.0f);
						dlogh = fminf(fmaxf(powf(rhoh30 / rhoh3, fpre * 0.3333333333333333f) - 1.f, -.1f), .1f);
						error = fabs(1.0f - rhoh3 / rhoh30);
						if (tid == 0) {
							h *= (1.f + w1 * dlogh);
						}
						if (last_dh * dlogh < 0.f) {
							w1 *= 0.5;
						} else {
							w1 = 1.f;
						}
						last_dh = dlogh;
					}
					__syncthreads();
					if (tid == 0) {
						if (iter > 100000) {
							PRINT("Solver failed to converge - %i %e %e %e\n", iter, h, dlogh, error);
							if (iter > 100010) {
								__trap();
							}
						}
					}
					for (int dim = 0; dim < NDIM; dim++) {
						if (self.outer_box.end[dim] < range_fixed(x[dim] + fixed32(h)) + range_fixed::min()) {
							box_xceeded = true;
							break;
						}
						if (range_fixed(x[dim]) < self.outer_box.begin[dim] + range_fixed(h) + range_fixed::min()) {
							box_xceeded = true;
							break;
						}
					}
					__syncthreads();
					iter++;
					shared_reduce_add<int, SMOOTHLEN_BLOCK_SIZE>(box_xceeded);
				} while (error > SPH_SMOOTHLEN_TOLER && !box_xceeded);
				if (!box_xceeded) {
					const float hinv = 1.f / h; // 4
					const float h2 = sqr(h);    // 1
					drho_dh = 0.f;
					float rhoh30 = (3.0f * data.N) / (4.0f * float(M_PI));
					for (int j = tid; j < ws.x.size(); j += block_size) {
						const float dx = distance(x[XDIM], ws.x[j]); // 2
						const float dy = distance(x[YDIM], ws.y[j]); // 2
						const float dz = distance(x[ZDIM], ws.z[j]); // 2
						const float r2 = sqr(dx, dy, dz);            // 2
						const float r = sqrt(r2);                    // 4
						const float q = r * hinv;                    // 1
						if (q < 1.f) {                               // 1
							const float w = kernelW(q); // 4
							const float dwdq = dkernelW_dq(q);
							drho_dh -= (3.f * w + q * dwdq);
						}
					}
					shared_reduce_add<float, SMOOTHLEN_BLOCK_SIZE>(drho_dh);
					drho_dh *= 0.33333333333f / rhoh30;
					const float fpre = 1.0f / (1.0f + drho_dh);
					if (tid == 0) {
						data.fpre_snk[snki] = fpre;
						//					data.converged_snk[snki] = true;
					}
					hmin = fminf(hmin, h);
					hmax = fmaxf(hmax, h);
				} else {
					if (tid == 0) {
						atomicAdd(&reduce->flag, 1);
					}
				}
			}
		}
		shared_reduce_add<int, SMOOTHLEN_BLOCK_SIZE>(flops);
		if (tid == 0) {
			atomicAdd(&reduce->flops, (double) flops);
			atomicMax(&reduce->hmax, hmax);
			atomicMin(&reduce->hmin, hmin);
			index = atomicAdd(&reduce->counter, 1);
		}
		__syncthreads();
	}
	(&ws)->~smoothlen_workspace();
}

__global__ void sph_cuda_mark_semiactive(sph_run_params params, sph_run_cuda_data data, sph_reduction* reduce) {
	const int tid = threadIdx.x;
	const int block_size = blockDim.x;
	__shared__
	int index;
	__shared__ mark_semiactive_workspace ws;
	if (tid == 0) {
		index = atomicAdd(&reduce->counter, 1);
	}
	__syncthreads();
	new (&ws) mark_semiactive_workspace();

	array<fixed32, NDIM> x;
	while (index < data.nselfs) {
		int flops = 0;
		ws.x.resize(0);
		ws.y.resize(0);
		ws.z.resize(0);
		ws.h.resize(0);
		ws.rungs.resize(0);
		const sph_tree_node& self = data.trees[data.selfs[index]];
		for (int ni = self.neighbor_range.first; ni < self.neighbor_range.second; ni++) {
			const sph_tree_node& other = data.trees[data.neighbors[ni]];
			const int maxpi = round_up(other.part_range.second - other.part_range.first, block_size) + other.part_range.first;
			for (int pi = other.part_range.first + tid; pi < maxpi; pi += block_size) {
				bool contains = false;
				int j;
				int total;
				float h;
				int rung;
				if (pi < other.part_range.second) {
					h = data.h[pi];
					rung = data.rungs[pi];
					if (rung >= params.min_rung) {
						x[XDIM] = data.x[pi];
						x[YDIM] = data.y[pi];
						x[ZDIM] = data.z[pi];
						contains = (self.outer_box.contains(x));
						if (!contains) {
							contains = true;
							for (int dim = 0; dim < NDIM; dim++) {
								if (distance(x[dim], self.inner_box.begin[dim]) + h < 0.f) {
									contains = false;
									break;
								}
								if (distance(self.inner_box.end[dim], x[dim]) + h < 0.f) {
									contains = false;
									break;
								}
							}
						}
					}
				}
				j = contains;
				compute_indices<MARK_SEMIACTIVE_BLOCK_SIZE>(j, total);
				const int offset = ws.x.size();
				__syncthreads();
				const int next_size = offset + total;
				ws.x.resize(next_size);
				ws.y.resize(next_size);
				ws.z.resize(next_size);
				ws.h.resize(next_size);
				ws.rungs.resize(next_size);
				if (contains) {
					const int k = offset + j;
					ws.x[k] = x[XDIM];
					ws.y[k] = x[YDIM];
					ws.z[k] = x[ZDIM];
					ws.h[k] = h;
					ws.rungs[k] = rung;
				}
			}
		}
		for (int i = self.part_range.first; i < self.part_range.second; i++) {
			__syncthreads();
			const int snki = self.sink_part_range.first - self.part_range.first + i;
			if (data.rungs[i] >= params.min_rung) {
				if (tid == 0) {
					data.sa_snk[snki] = true;
				}
			} else {
				const auto x0 = data.x[i];
				const auto y0 = data.y[i];
				const auto z0 = data.z[i];
				const auto h0 = data.h[i];
				const auto h02 = sqr(h0);
				int semiactive = 0;
				const int jmax = round_up(ws.x.size(), block_size);
				if (tid == 0) {
					data.sa_snk[snki] = false;
				}
				for (int j = tid; j < jmax; j += block_size) {
					if (j < ws.x.size()) {
						const auto x1 = ws.x[j];
						const auto y1 = ws.y[j];
						const auto z1 = ws.z[j];
						const auto h1 = ws.h[j];
						const auto h12 = sqr(h1);
						const float dx = distance(x0, x1);
						const float dy = distance(y0, y1);
						const float dz = distance(z0, z1);
						const float r2 = sqr(dx, dy, dz);
						if (r2 < fmaxf(h02, h12)) {
							semiactive++;
						}
					}
					shared_reduce_add<int, MARK_SEMIACTIVE_BLOCK_SIZE>(semiactive);
					if (semiactive) {
						if (tid == 0) {
							data.sa_snk[snki] = true;
						}
						break;
					}
				}
			}
		}
		shared_reduce_add<int, MARK_SEMIACTIVE_BLOCK_SIZE>(flops);
		if (tid == 0) {
			index = atomicAdd(&reduce->counter, 1);
		}
		__syncthreads();
	}
	(&ws)->~mark_semiactive_workspace();

}

__global__ void sph_cuda_hydro(sph_run_params params, sph_run_cuda_data data, sph_reduction* reduce) {
	const int tid = threadIdx.x;
	const int block_size = blockDim.x;
	__shared__
	int index;
	__shared__ hydro_workspace ws;
	new (&ws) hydro_workspace();
	if (tid == 0) {
		index = atomicAdd(&reduce->counter, 1);
	}
	__syncthreads();
	array<fixed32, NDIM> x;
	float total_vsig_max = 0.;
	int max_rung_hydro = 0;
	int max_rung_grav = 0;
	int max_rung = 0;
	int flops = 0;
	const float ainv = 1.0f / params.a;
	while (index < data.nselfs) {
		const sph_tree_node& self = data.trees[data.selfs[index]];
		ws.rec1.resize(0);
		ws.rec2.resize(0);
		for (int ni = self.neighbor_range.first; ni < self.neighbor_range.second; ni++) {
			const sph_tree_node& other = data.trees[data.neighbors[ni]];
			const int maxpi = round_up(other.part_range.second - other.part_range.first, block_size) + other.part_range.first;
			for (int pi = other.part_range.first + tid; pi < maxpi; pi += block_size) {
				bool contains = false;
				int j;
				int total;
				if (pi < other.part_range.second) {
					x[XDIM] = data.x[pi];
					x[YDIM] = data.y[pi];
					x[ZDIM] = data.z[pi];
					if (self.outer_box.contains(x)) {
						contains = true;
					}
					if (!contains) {
						contains = true;
						const float& h = data.h[pi];
						for (int dim = 0; dim < NDIM; dim++) {
							if (distance(x[dim], self.inner_box.begin[dim]) + h < 0.f) {
								contains = false;
								break;
							}
							if (distance(self.inner_box.end[dim], x[dim]) + h < 0.f) {
								contains = false;
								break;
							}
						}
					}
				}
				j = contains;
				compute_indices<HYDRO_BLOCK_SIZE>(j, total);
				const int offset = ws.rec1.size();
				__syncthreads();
				const int next_size = offset + total;
				ws.rec1.resize(next_size);
				ws.rec2.resize(next_size);
				if (contains) {
					const int k = offset + j;
					ws.rec1[k].x = x[XDIM];
					ws.rec1[k].y = x[YDIM];
					ws.rec1[k].z = x[ZDIM];
					ws.rec1[k].h = data.h[pi];
					ws.rec2[k].vx = data.vx[pi];
					ws.rec2[k].vy = data.vy[pi];
					ws.rec2[k].vz = data.vz[pi];
					ws.rec2[k].entr = data.entr[pi];
					ws.rec2[k].alpha = data.alpha[pi];
					ws.rec2[k].fpre = data.fpre[pi];
					ws.rec2[k].rung = data.rungs[pi];
					if (params.stars) {
						ws.rec2[k].cold_frac = data.cold_frac[pi];
					} else {
						ws.rec2[k].cold_frac = 0.f;
					}
					if (params.diffusion) {
						ws.rec2[k].shearv = data.shearv[pi];
						if (data.chemistry) {
							ws.rec2[k].chem = data.chem[pi];
						}
					}
				}
			}
		}
		for (int i = self.part_range.first; i < self.part_range.second; i++) {
			__syncthreads();
			int rung_i = data.rungs[i];
			bool use = rung_i >= params.min_rung;
			const int snki = self.sink_part_range.first - self.part_range.first + i;
			const float m = data.m;
			const float minv = 1.f / m;
			const float c0 = float(3.0f / 4.0f / M_PI * data.N);
			const float c0inv = 1.0f / c0;
			if (use) {
				const auto x_i = data.x[i];
				const auto y_i = data.y[i];
				const auto z_i = data.z[i];
				const auto vx_i = data.vx[i];
				const auto vy_i = data.vy[i];
				const auto vz_i = data.vz[i];
				const float h_i = data.h[i];
				const float h2_i = sqr(h_i);
				const float hinv_i = 1.f / h_i;
				const float h3inv_i = (sqr(hinv_i) * hinv_i);
				const float rho_i = m * c0 * h3inv_i;
				const float rhoinv_i = minv * c0inv * sqr(h_i) * h_i;
				const float K_i = data.entr[i];
				const float alpha_i = data.alpha[i];
				const float gamma0 = data.def_gamma;
				const float p_i = K_i * powf(rho_i, gamma0);
				float cfrac_i;
				if (params.stars) {
					cfrac_i = data.cold_frac[i];
				} else {
					cfrac_i = 0.f;
				}
				const float hfrac_i = 1.f - cfrac_i;
				const float c_i = sqrtf(gamma0 * p_i * rhoinv_i);
				const float fpre_i = data.fpre[i];
				//	float shearv_i;
				//	array<float, NCHEMFRACS> frac_i;
				/*	if (params.diffusion) {
				 shearv_i = data.shearv[i];
				 if (data.chemistry) {
				 frac_i = data.chem[i];
				 }
				 }*/
				float ax = 0.f;
				float ay = 0.f;
				float az = 0.f;
				float de_dt = 0.f;
				float dcm_dt = 0.f;
				//	array<float, NCHEMFRACS> dfrac_dt;
				/*		if (params.diffusion && data.chemistry) {
				 for (int fi = 0; fi < NCHEMFRACS; fi++) {
				 dfrac_dt[fi] = 0.f;
				 }
				 }*/
				float dtinv_cfl = 0.f;
				float one = 0.0f;
				constexpr float tiny = 1e-30f;
				float vsig = 0.0f;
				const float& adot = params.adot;
				for (int j = tid; j < ws.rec1.size(); j += block_size) {
					const auto rec1 = ws.rec1[j];
					const auto rec2 = ws.rec2[j];
					const float h_j = rec1.h;
					const float hinv_j = 1.f / h_j;															// 4
					const fixed32 x_j = rec1.x;
					const fixed32 y_j = rec1.y;
					const fixed32 z_j = rec1.z;
					const float x_ij = distance(x_i, x_j);				// 2
					const float y_ij = distance(y_i, y_j);				// 2
					const float z_ij = distance(z_i, z_j);				// 2
					const float r2 = sqr(x_ij, y_ij, z_ij);
					const float r = sqrt(r2);
					const float q_i = r * hinv_i;								// 1
					const float q_j = r * hinv_j;
					if (q_i < 1.f || q_j < 1.f) {
						const float cfrac_j = rec2.cold_frac;
						const float hfrac_j = 1.f - cfrac_j;
						const float vx_j = rec2.vx;
						const float vy_j = rec2.vy;
						const float vz_j = rec2.vz;
						const float fpre_j = rec2.fpre;
						const float h2_j = sqr(h_j);
						const float h3inv_j = sqr(hinv_j) * hinv_j;
						const float rho_j = m * c0 * h3inv_j;													// 2
						const float rhoinv_j = minv * c0inv * sqr(h_j) * h_j;								// 5
						const float K_j = rec2.entr;
						const float p_j = K_j * powf(rho_j, gamma0);
						const float c_j = sqrtf(gamma0 * p_j * rhoinv_j);									// 6
						const float alpha_j = rec2.alpha;
						const float vx0_ij = vx_i - vx_j;
						const float vy0_ij = vy_i - vy_j;
						const float vz0_ij = vz_i - vz_j;
						const float vx_ij = vx0_ij + x_ij * adot;
						const float vy_ij = vy0_ij + y_ij * adot;
						const float vz_ij = vz0_ij + z_ij * adot;
						const float rinv = 1.0f / (r + tiny);
						const float vdotx_ij = fminf(0.0f, x_ij * vx_ij + y_ij * vy_ij + z_ij * vz_ij);
						const float h_ij = 0.5f * (h_i + h_j);
						const float mu_ij = vdotx_ij * h_ij / (r2 + 0.01f * sqr(h_ij));
						const float rho_ij = 0.5f * (rho_i + rho_j);
						const float c_ij = 0.5f * (c_i + c_j);
						const float alpha_ij = 0.5f * (alpha_i + alpha_j);
						const float vsig_ij = alpha_ij * (c_ij - params.beta * mu_ij);
						const float pi_ij = -mu_ij * vsig_ij / rho_ij;
						const float dWdr_i = fpre_i * dkernelW_dq(q_i) * hinv_i * h3inv_i;
						const float dWdr_j = fpre_j * dkernelW_dq(q_j) * hinv_j * h3inv_j;
						const float dWdr_ij = 0.5f * (dWdr_i + dWdr_j);
						const float dWdr_x_ij = x_ij * rinv * dWdr_ij;
						const float dWdr_y_ij = y_ij * rinv * dWdr_ij;
						const float dWdr_z_ij = z_ij * rinv * dWdr_ij;
						const float dWdr_x_i = x_ij * rinv * dWdr_i;
						const float dWdr_y_i = y_ij * rinv * dWdr_i;
						const float dWdr_z_i = z_ij * rinv * dWdr_i;
						const float dWdr_x_j = x_ij * rinv * dWdr_j;
						const float dWdr_y_j = y_ij * rinv * dWdr_j;
						const float dWdr_z_j = z_ij * rinv * dWdr_j;
						const float dp_i = p_i * sqr(rhoinv_i);
						const float dp_j = p_j * sqr(rhoinv_j);
						one += m / rho_i * kernelW(q_i) * h3inv_i;
						ax -= m * ainv * (dp_i * dWdr_x_i + dp_j * dWdr_x_j);
						ay -= m * ainv * (dp_i * dWdr_y_i + dp_j * dWdr_y_j);
						az -= m * ainv * (dp_i * dWdr_z_i + dp_j * dWdr_z_j);
						ax -= m * ainv * (pi_ij * dWdr_x_ij);
						ay -= m * ainv * (pi_ij * dWdr_y_ij);
						az -= m * ainv * (pi_ij * dWdr_z_ij);
						de_dt += (gamma0 - 1.f) * powf(rho_i, 1.f - gamma0) * 0.5f * m * ainv * pi_ij * (vx_ij * dWdr_x_ij + vy_ij * dWdr_y_ij + vz_ij * dWdr_z_ij);
						if (params.phase == 1 || params.damping > 0.f) {
							vsig = fmaxf(vsig, vsig_ij);
						}
						if (params.phase == 1) {
							float dtinv = (c_ij + 0.6f * vsig_ij) / fminf(h_i, 2.f * h_j);
							dtinv_cfl = fmaxf(dtinv_cfl, dtinv);
						}
					}
				}
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(de_dt);
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(ax);
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(ay);
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(az);
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(one);
				if (params.phase == 1) {
					shared_reduce_max<HYDRO_BLOCK_SIZE>(dtinv_cfl);
					shared_reduce_max<HYDRO_BLOCK_SIZE>(vsig);
				}
				if (params.stars) {
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(dcm_dt);
				}

				//		ALWAYS_ASSERT(data.converged_snk[snki] == 0);
				if (fabs(1. - one) > 1.0e-4 && tid == 0) {
					PRINT("one is off %e %i\n", one, data.converged_snk[snki]);
					__trap();
				}
				if (tid == 0) {
					float gx_i;
					float gy_i;
					float gz_i;
					if (data.gravity) {
						gx_i = data.gx_snk[snki];
						gy_i = data.gy_snk[snki];
						gz_i = data.gz_snk[snki];
					} else {
						gx_i = 0.f;
						gy_i = 0.f;
						gz_i = 0.f;
					}
					ax += gx_i;
					ay += gy_i;
					az += gz_i;
					data.dvx_con[snki] = ax;
					data.dvy_con[snki] = ay;
					data.dvz_con[snki] = az;
					data.dentr_con[snki] = de_dt;
					if (params.diffusion && data.chemistry) {
						for (int fi = 0; fi < NCHEMFRACS; fi++) {
							data.dchem_con[snki][fi] = 0.f;
						}
					}
					if (params.phase == 1) {
						const float divv = data.divv_snk[snki];
						//				const float dtinv_divv = params.a * fabsf(divv - 3.f * params.adot * ainv) * (1.f / 3.f);
						float dtinv_hydro1 = 1.0e-30f;
						//			dtinv_hydro1 = fmaxf(dtinv_hydro1, dtinv_divv);
						dtinv_hydro1 = fmaxf(dtinv_hydro1, dtinv_cfl);
						const float a2_1 = sqr(ax, ay, az);
						const float a2_2 = sqr(ax - gx_i, ay - gy_i, az - gz_i);
						const float a2 = fminf(a2_1, a2_2);
						const float dtinv_acc = sqrtf(sqrtf(a2) * hinv_i);
						const float dtinv_hydro2 = dtinv_acc;
						float dthydro = params.cfl * params.a / (dtinv_hydro1 + 1e-30f);
						dthydro = fminf(data.eta * sqrtf(params.a) / (dtinv_hydro2 + 1e-30f), dthydro);
						const float g2 = sqr(gx_i, gy_i, gz_i);
						const float dtinv_grav = sqrtf(sqrtf(g2) * hinv_i);
						float dtgrav = data.eta * sqrtf(params.a) / (dtinv_grav + 1e-30f);
						dthydro = fminf(dthydro, params.max_dt);
						dtgrav = fminf(dtgrav, params.max_dt);
						total_vsig_max = fmaxf(total_vsig_max, dtinv_hydro1 * h_i);
						char& rung = data.rungs[i];
						data.oldrung_snk[snki] = rung;
						const int rung_hydro = ceilf(log2f(params.t0) - log2f(dthydro));
						const int rung_grav = ceilf(log2f(params.t0) - log2f(dtgrav));
						max_rung_hydro = max(max_rung_hydro, rung_hydro);
						max_rung_grav = max(max_rung_grav, rung_grav);
						rung = max(max((int) max(rung_hydro, rung_grav), max(params.min_rung, (int) rung - 1)), 1);
						max_rung = max(max_rung, rung);
						if (rung < 0 || rung >= MAX_RUNG) {
							if (tid == 0) {
								PRINT("Rung out of range \n");
								__trap();
							}
						}
					}
				}
			}
		}
		shared_reduce_add<int, HYDRO_BLOCK_SIZE>(flops);
		if (tid == 0) {
			atomicAdd(&reduce->flops, (float) flops);
			index = atomicAdd(&reduce->counter, 1);
		}
		flops = 0;
		__syncthreads();
	}
	if (tid == 0 && params.phase == 1) {
		atomicMax(&reduce->vsig_max, total_vsig_max);
		atomicMax(&reduce->max_rung, max_rung);
		atomicMax(&reduce->max_rung_hydro, max_rung_hydro);
		atomicMax(&reduce->max_rung_grav, max_rung_grav);
	}
	(&ws)->~hydro_workspace();
}

__global__ void sph_cuda_aux(sph_run_params params, sph_run_cuda_data data, sph_reduction* reduce) {
	const int tid = threadIdx.x;
	const int block_size = blockDim.x;
	__shared__
	int index;
	__shared__ aux_workspace ws;
	if (tid == 0) {
		index = atomicAdd(&reduce->counter, 1);
	}
	__syncthreads();
	array<fixed32, NDIM> x;
	int flops = 0;
	new (&ws) aux_workspace();

	const double kb = (double) constants::kb * sqr((double) params.code_to_s) / ((double) params.code_to_g * sqr((double) params.code_to_cm));
	const double mh = (double) constants::mh / (double) params.code_to_g;
	const float cv0 = kb / mh;
	const float ainv = 1.0f / params.a;
	while (index < data.nselfs) {
		const sph_tree_node& self = data.trees[data.selfs[index]];
		ws.rec1.resize(0);
		ws.rec2.resize(0);
		for (int ni = self.neighbor_range.first; ni < self.neighbor_range.second; ni++) {
			const sph_tree_node& other = data.trees[data.neighbors[ni]];
			const int maxpi = round_up(other.part_range.second - other.part_range.first, block_size) + other.part_range.first;
			for (int pi = other.part_range.first + tid; pi < maxpi; pi += block_size) {
				bool contains = false;
				int j;
				int total;
				if (pi < other.part_range.second) {
					x[XDIM] = data.x[pi];
					x[YDIM] = data.y[pi];
					x[ZDIM] = data.z[pi];
					contains = self.outer_box.contains(x);
				}
				j = contains;
				compute_indices<AUX_BLOCK_SIZE>(j, total);
				const int offset = ws.rec1.size();
				__syncthreads();
				const int next_size = offset + total;
				ws.rec1.resize(next_size);
				ws.rec2.resize(next_size);
				if (contains) {
					const int k = offset + j;
					ws.rec1[k].x = x[XDIM];
					ws.rec1[k].y = x[YDIM];
					ws.rec1[k].z = x[ZDIM];
					ws.rec2[k].vx = data.vx[pi];
					ws.rec2[k].vy = data.vy[pi];
					ws.rec2[k].vz = data.vz[pi];
					if (data.chemistry) {
						ws.rec2[k].gamma = data.gamma[pi];
					} else {
						ws.rec2[k].gamma = data.def_gamma;
					}
					ws.rec2[k].h = data.h[pi];
					ws.rec2[k].entr = data.entr[pi];
				}
			}
		}
		for (int i = self.part_range.first; i < self.part_range.second; i++) {
			__syncthreads();
			const int snki = self.sink_part_range.first - self.part_range.first + i;
			int rung_i = data.rungs[i];
			const bool active = rung_i >= params.min_rung;
			const bool use = active;
			const float m = data.m;
			const float minv = 1.f / m;
			const float c0 = float(3.0f / 4.0f / M_PI * data.N);
			const float c0inv = 1.0f / c0;
			if (use) {
				const auto x_i = data.x[i];
				const auto y_i = data.y[i];
				const auto z_i = data.z[i];
				const auto vx_i = data.vx[i];
				const auto vy_i = data.vy[i];
				const auto vz_i = data.vz[i];
				float h_i;
				h_i = data.h[i];
				const float h2_i = sqr(h_i);
				const float hinv_i = 1.f / h_i;
				const float h3inv_i = (sqr(hinv_i) * hinv_i);
				const float rho_i = m * c0 * h3inv_i;
				const float rhoinv_i = minv * c0inv * sqr(h_i) * h_i;
				const float fpre_i = data.fpre_snk[snki];
				float K_i, p_i, c_i;
				const float gamma0 = data.def_gamma;
				K_i = data.entr[i];
				p_i = K_i * powf(rho_i, gamma0);
				c_i = sqrtf(gamma0 * p_i * rhoinv_i);
				float dvx_dx = 0.0f;
				float dvx_dy = 0.0f;
				float dvx_dz = 0.0f;
				float dvy_dx = 0.0f;
				float dvy_dy = 0.0f;
				float dvy_dz = 0.0f;
				float dvz_dx = 0.0f;
				float dvz_dy = 0.0f;
				float dvz_dz = 0.0f;
				float vsig = 0.f;
				const float adot = params.adot;
				for (int j = tid; j < ws.rec1.size(); j += block_size) {
					const auto rec1 = ws.rec1[j];
					const auto rec2 = ws.rec2[j];
					const float vx_j = rec2.vx;
					const float vy_j = rec2.vy;
					const float vz_j = rec2.vz;
					const fixed32 x_j = rec1.x;
					const fixed32 y_j = rec1.y;
					const fixed32 z_j = rec1.z;
					const float x_ij = distance(x_i, x_j);				// 2
					const float y_ij = distance(y_i, y_j);				// 2
					const float z_ij = distance(z_i, z_j);				// 2
					const float vx0_ij = vx_i - vx_j;
					const float vy0_ij = vy_i - vy_j;
					const float vz0_ij = vz_i - vz_j;
					const float vx_ij = vx0_ij + x_ij * adot;
					const float vy_ij = vy0_ij + y_ij * adot;
					const float vz_ij = vz0_ij + z_ij * adot;
					const float r2 = sqr(x_ij, y_ij, z_ij);
					const float r = sqrt(r2);
					const float rinv = 1.0f / (1.0e-30f + r);
					const float q_i = r * hinv_i;
					const float dWdr_i = fpre_i * dkernelW_dq(q_i) * hinv_i * h3inv_i;
					const float dWdr_x_i = dWdr_i * rinv * x_ij;
					const float dWdr_y_i = dWdr_i * rinv * y_ij;
					const float dWdr_z_i = dWdr_i * rinv * z_ij;
					const float vdotx_ij = fminf(vx_ij * x_ij + vy_ij * y_ij + vz_ij * z_ij, 0.f);
					const float h_j = rec2.h;
					const float hinv_j = 1.f / h_j;
					const float h3inv_j = (sqr(hinv_j) * hinv_j);
					const float rho_j = m * c0 * h3inv_j;
					const float gamma_j = rec2.gamma;
					const float K_j = rec2.entr;
					const float p_j = K_j * powf(rho_j, gamma0);
					const float c_j = sqrtf(gamma_j * p_j / rho_j);
					const float w_ij = vdotx_ij * rinv;
					const float vsig_i = 0.5f * (c_i + c_j) - w_ij;
					dvx_dx -= vx_ij * dWdr_x_i;
					dvy_dx -= vy_ij * dWdr_x_i;
					dvz_dx -= vz_ij * dWdr_x_i;
					dvx_dy -= vx_ij * dWdr_y_i;
					dvy_dy -= vy_ij * dWdr_y_i;
					dvz_dy -= vz_ij * dWdr_y_i;
					dvx_dz -= vx_ij * dWdr_z_i;
					dvy_dz -= vy_ij * dWdr_z_i;
					dvz_dz -= vz_ij * dWdr_z_i;
					vsig = fmaxf(vsig, vsig_i);
				}
				const float mrhoinvainv_i = m * rhoinv_i * ainv;
				dvx_dx *= mrhoinvainv_i;
				dvx_dy *= mrhoinvainv_i;
				dvx_dz *= mrhoinvainv_i;
				dvy_dx *= mrhoinvainv_i;
				dvy_dy *= mrhoinvainv_i;
				dvy_dz *= mrhoinvainv_i;
				dvz_dx *= mrhoinvainv_i;
				dvz_dy *= mrhoinvainv_i;
				dvz_dz *= mrhoinvainv_i;
				float shear_xx, shear_xy, shear_xz, shear_yy, shear_yz, shear_zz;
				float div_v, curl_vx, curl_vy, curl_vz;
				div_v = dvx_dx + dvy_dy + dvz_dz;
				curl_vx = dvz_dy - dvy_dz;
				curl_vy = -dvz_dx + dvx_dz;
				curl_vz = dvy_dx - dvx_dy;
				shear_xx = dvx_dx - (1.f / 3.f) * div_v;
				shear_yy = dvy_dy - (1.f / 3.f) * div_v;
				shear_zz = dvz_dz - (1.f / 3.f) * div_v;
				shear_xy = 0.5f * (dvx_dy + dvy_dx);
				shear_xz = 0.5f * (dvx_dz + dvz_dx);
				shear_yz = 0.5f * (dvy_dz + dvz_dy);
				shared_reduce_add<float, AUX_BLOCK_SIZE>(shear_xx);
				shared_reduce_add<float, AUX_BLOCK_SIZE>(shear_xy);
				shared_reduce_add<float, AUX_BLOCK_SIZE>(shear_xz);
				shared_reduce_add<float, AUX_BLOCK_SIZE>(shear_yy);
				shared_reduce_add<float, AUX_BLOCK_SIZE>(shear_yz);
				shared_reduce_add<float, AUX_BLOCK_SIZE>(shear_zz);
				shared_reduce_add<float, AUX_BLOCK_SIZE>(div_v);
				shared_reduce_add<float, AUX_BLOCK_SIZE>(curl_vx);
				shared_reduce_add<float, AUX_BLOCK_SIZE>(curl_vy);
				shared_reduce_add<float, AUX_BLOCK_SIZE>(curl_vz);
				shared_reduce_max<AUX_BLOCK_SIZE>(vsig);
				if (tid == 0) {
					const float shearv = sqrtf(sqr(shear_xx) + sqr(shear_yy) + sqr(shear_zz) + 2.0f * (sqr(shear_xy) + sqr(shear_yz) + sqr(shear_xz)));
					const float curlv = sqrtf(sqr(curl_vx, curl_vy, curl_vz));
					data.divv_snk[snki] = div_v;
					data.shearv_snk[snki] = shearv;
					/* float& alpha = data.alpha_snk[snki];
					 const float dt = params.t0 * rung_dt[rung_i];
					 const float limiter = fabs(div_v) / (fabs(div_v) + curlv + 1e-30f);
					 const float S = fmaxf(0.f, -div_v) * limiter;
					 const float tauinv = params.alpha_decay * c_i * hinv_i * ainv;
					 alpha = (alpha + (params.alpha1 * S + params.alpha0 * tauinv) * dt) / (1.f + dt * (params.alpha0 * S + tauinv));*/
					float& alpha = data.alpha_snk[snki];
					const float divv0 = params.tau > 0.f ? data.divv0_snk[snki] : div_v;
					data.divv0_snk[snki] = div_v;
					if (params.tau > 0.f) {
						const float dt = params.t0 * rung_dt[rung_i];
						const float ddivv_dt = (div_v - divv0) / dt - 0.5f * params.adot * ainv * (div_v + divv0);
						const float S = sqr(h_i) * fmaxf(0.f, -ddivv_dt) * sqr(params.a);
						const float limiter = sqr(div_v) / (sqr(div_v) + sqr(curlv) + 1.0e-4f * sqr(c_i / h_i * ainv));
						const float alpha_targ = S / (S + sqr(c_i));
						const float lambda0 = params.alpha_decay * vsig * hinv_i * ainv * dt;
						if (alpha < limiter * alpha_targ) {
							alpha = limiter * alpha_targ;
						} else {
							alpha = (alpha + lambda0 * limiter * alpha_targ) / (1.f + lambda0);
						}
					} else {
						alpha = 0.f;
					}

					//				float& alpha = data.alpha_snk[snki];
					//				alpha = 0.75f;
				}
			}
		}
		shared_reduce_add<int, AUX_BLOCK_SIZE>(flops);
		if (tid == 0) {
			atomicAdd(&reduce->flops, (float) flops);
			index = atomicAdd(&reduce->counter, 1);
		}
		flops = 0;
		__syncthreads();
	}
	(&ws)->~aux_workspace();
}

__global__ void sph_cuda_rungs(sph_run_params params, sph_run_cuda_data data, sph_reduction* reduce) {
	const int tid = threadIdx.x;
	const int block_size = blockDim.x;
	__shared__
	int index;
	__shared__ rungs_workspace ws;
	if (tid == 0) {
		index = atomicAdd(&reduce->counter, 1);
	}
	__syncthreads();
	new (&ws) rungs_workspace();

	array<fixed32, NDIM> x;
	int changes = 0;
	while (index < data.nselfs) {
		int flops = 0;
		ws.x.resize(0);
		ws.y.resize(0);
		ws.z.resize(0);
		ws.h.resize(0);
		ws.rungs.resize(0);
		const sph_tree_node& self = data.trees[data.selfs[index]];
		for (int ni = self.neighbor_range.first; ni < self.neighbor_range.second; ni++) {
			const sph_tree_node& other = data.trees[data.neighbors[ni]];
			const int maxpi = round_up(other.part_range.second - other.part_range.first, block_size) + other.part_range.first;
			for (int pi = other.part_range.first + tid; pi < maxpi; pi += block_size) {
				bool contains = false;
				int j;
				int total;
				float h;
				int rung;
				if (pi < other.part_range.second) {
					h = data.h[pi];
					rung = data.rungs[pi];
					if (rung >= params.min_rung) {
						x[XDIM] = data.x[pi];
						x[YDIM] = data.y[pi];
						x[ZDIM] = data.z[pi];
						if (self.outer_box.contains(x)) {
							contains = true;
						}
						if (!contains) {
							contains = true;
							for (int dim = 0; dim < NDIM; dim++) {
								if (distance(x[dim], self.inner_box.begin[dim]) + h < 0.f) {
									contains = false;
									break;
								}
								if (distance(self.inner_box.end[dim], x[dim]) + h < 0.f) {
									contains = false;
									break;
								}
							}
						}
					}
				}
				j = contains;
				compute_indices<RUNGS_BLOCK_SIZE>(j, total);
				const int offset = ws.x.size();
				__syncthreads();
				const int next_size = offset + total;
				ws.x.resize(next_size);
				ws.y.resize(next_size);
				ws.z.resize(next_size);
				ws.h.resize(next_size);
				ws.rungs.resize(next_size);
				if (contains) {
					ASSERT(j < total);
					const int k = offset + j;
					ws.x[k] = x[XDIM];
					ws.y[k] = x[YDIM];
					ws.z[k] = x[ZDIM];
					ws.h[k] = h;
					ws.rungs[k] = rung;
				}
			}
		}
		for (int i = self.part_range.first; i < self.part_range.second; i++) {
			__syncthreads();
			if (data.rungs[i] >= params.min_rung) {
				const auto x_i = data.x[i];
				const auto y_i = data.y[i];
				const auto z_i = data.z[i];
				const auto h_i = data.h[i];
				char& rung_i = data.rungs[i];
				const auto h2_i = sqr(h_i);
				const int jmax = round_up(ws.x.size(), block_size);
				int max_rung_j = 0;
				for (int j = tid; j < jmax; j += block_size) {
					if (j < ws.x.size()) {
						const auto x_j = ws.x[j];
						const auto y_j = ws.y[j];
						const auto z_j = ws.z[j];
						const auto h_j = ws.h[j];
						const int rung_j = ws.rungs[j];
						const auto h2_j = sqr(h_j);
						const float x_ij = distance(x_i, x_j);
						const float y_ij = distance(y_i, y_j);
						const float z_ij = distance(z_i, z_j);
						const float r2 = sqr(x_ij, y_ij, z_ij);
						if (r2 < fmaxf(h2_i, h2_j)) {
							max_rung_j = max(max_rung_j, rung_j);
						}
					}
				}
				shared_reduce_max<RUNGS_BLOCK_SIZE>(max_rung_j);
				if (tid == 0) {
					if (rung_i < max_rung_j - 1) {
						changes++;
						rung_i = max_rung_j - 1;
					}
				}
			}
		}
		shared_reduce_add<int, RUNGS_BLOCK_SIZE>(flops);
		if (tid == 0) {
			index = atomicAdd(&reduce->counter, 1);
		}
		__syncthreads();
	}
	if (tid == 0) {
		atomicAdd(&reduce->flag, changes);
	}
	(&ws)->~rungs_workspace();

}

sph_run_return sph_run_cuda(sph_run_params params, sph_run_cuda_data data, hipStream_t stream) {
	timer tm;
	sph_run_return rc;
	sph_reduction* reduce;
	CUDA_CHECK(hipMallocManaged(&reduce, sizeof(sph_reduction)));
	reduce->counter = reduce->flag = 0;
	reduce->hmin = std::numeric_limits<float>::max();
	reduce->hmax = 0.0f;
	reduce->flops = 0.0;
	reduce->vsig_max = 0.0;
	reduce->max_rung_grav = 0;
	reduce->max_rung_hydro = 0;
	reduce->max_rung = 0;
	static int smoothlen_nblocks;
	static int semiactive_nblocks;
	static int hydro_nblocks;
	static int aux_nblocks;
	static int xsph_nblocks;
	static int rungs_nblocks;
	static bool first = true;
	if (first) {
		first = false;
		CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&smoothlen_nblocks, (const void*) sph_cuda_smoothlen, SMOOTHLEN_BLOCK_SIZE, 0));
		smoothlen_nblocks *= cuda_smp_count();
		CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&semiactive_nblocks, (const void*) sph_cuda_mark_semiactive, MARK_SEMIACTIVE_BLOCK_SIZE, 0));
		semiactive_nblocks *= cuda_smp_count();
		CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&hydro_nblocks, (const void*) sph_cuda_hydro, HYDRO_BLOCK_SIZE, 0));
		hydro_nblocks *= cuda_smp_count();
		CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&aux_nblocks, (const void*) sph_cuda_aux, AUX_BLOCK_SIZE, 0));
		aux_nblocks *= cuda_smp_count();
		CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&xsph_nblocks, (const void*) sph_cuda_xsph, XSPH_BLOCK_SIZE, 0));
		xsph_nblocks *= cuda_smp_count();
		CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&rungs_nblocks, (const void*) sph_cuda_rungs, RUNGS_BLOCK_SIZE, 0));
		rungs_nblocks *= cuda_smp_count();
	}
	switch (params.run_type) {
	case SPH_RUN_SMOOTHLEN: {
		sph_cuda_smoothlen<<<smoothlen_nblocks, SMOOTHLEN_BLOCK_SIZE,0,stream>>>(params,data,reduce);
		cuda_stream_synchronize(stream);
		rc.rc = reduce->flag;
		rc.hmin = reduce->hmin;
		rc.hmax = reduce->hmax;
	}
	break;
	case SPH_RUN_MARK_SEMIACTIVE: {
		sph_cuda_mark_semiactive<<<semiactive_nblocks, MARK_SEMIACTIVE_BLOCK_SIZE,0,stream>>>(params,data,reduce);
		cuda_stream_synchronize(stream);
	}
	break;
	case SPH_RUN_RUNGS: {
		timer tm;
		tm.start();
		sph_cuda_rungs<<<hydro_nblocks, RUNGS_BLOCK_SIZE,0,stream>>>(params,data,reduce);
		cuda_stream_synchronize(stream);
		tm.stop();
		auto gflops = reduce->flops / tm.read() / (1024.0*1024*1024);
		rc.max_vsig = reduce->vsig_max;
		rc.max_rung_grav = reduce->max_rung_grav;
		rc.max_rung_hydro = reduce->max_rung_hydro;
		rc.max_rung = reduce->max_rung;
		rc.rc = reduce->flag;
	}
	break;
	case SPH_RUN_HYDRO: {
		sph_cuda_hydro<<<hydro_nblocks, HYDRO_BLOCK_SIZE,0,stream>>>(params,data,reduce);
		cuda_stream_synchronize(stream);
		rc.max_vsig = reduce->vsig_max;
		rc.max_rung_grav = reduce->max_rung_grav;
		rc.max_rung_hydro = reduce->max_rung_hydro;
		rc.max_rung = reduce->max_rung;
	}
	break;
	case SPH_RUN_AUX: {
		sph_cuda_aux<<<aux_nblocks, AUX_BLOCK_SIZE,0,stream>>>(params,data,reduce);
		cuda_stream_synchronize(stream);
	}
	break;
	case SPH_RUN_XSPH: {
		timer tm;
		tm.start();
		sph_cuda_xsph<<<xsph_nblocks, XSPH_BLOCK_SIZE,0,stream>>>(params,data,reduce);
		cuda_stream_synchronize(stream);
		tm.stop();
		PRINT( "XSPH time = %e\n", tm.read());
	}
	break;
}
	(hipFree(reduce));
	return rc;
}
