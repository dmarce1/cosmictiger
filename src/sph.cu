#include "hip/hip_runtime.h"
/*
 CosmicTiger - A cosmological N-Body code
 Copyright (C) 2021  Dominic C. Marcello

 This program is free software; you can redistribute it and/or
 modify it under the terms of the GNU General Public License
 as published by the Free Software Foundation; either version 2
 of the License, or (at your option) any later version.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program; if not, write to the Free Software
 Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 */

struct smoothlen_shmem {
	int index;
};

#include <cosmictiger/sph_cuda.hpp>
#include <cosmictiger/cuda_reduce.hpp>
#include <cosmictiger/constants.hpp>
#include <cosmictiger/timer.hpp>
#include <cosmictiger/kernel.hpp>

static __constant__ float rung_dt[MAX_RUNG] = { 1.0 / (1 << 0), 1.0 / (1 << 1), 1.0 / (1 << 2), 1.0 / (1 << 3), 1.0 / (1 << 4), 1.0 / (1 << 5), 1.0 / (1 << 6),
		1.0 / (1 << 7), 1.0 / (1 << 8), 1.0 / (1 << 9), 1.0 / (1 << 10), 1.0 / (1 << 11), 1.0 / (1 << 12), 1.0 / (1 << 13), 1.0 / (1 << 14), 1.0 / (1 << 15), 1.0
				/ (1 << 16), 1.0 / (1 << 17), 1.0 / (1 << 18), 1.0 / (1 << 19), 1.0 / (1 << 20), 1.0 / (1 << 21), 1.0 / (1 << 22), 1.0 / (1 << 23), 1.0 / (1 << 24),
		1.0 / (1 << 25), 1.0 / (1 << 26), 1.0 / (1 << 27), 1.0 / (1 << 28), 1.0 / (1 << 29), 1.0 / (1 << 30), 1.0 / (1 << 31) };

#define WORKSPACE_SIZE (160*1024)
#define HYDRO_SIZE (16*1024)

struct smoothlen_workspace {
	fixedcapvec<fixed32, WORKSPACE_SIZE> x;
	fixedcapvec<fixed32, WORKSPACE_SIZE> y;
	fixedcapvec<fixed32, WORKSPACE_SIZE> z;
};

struct mark_semiactive_workspace {
	fixedcapvec<fixed32, WORKSPACE_SIZE + 1> x;
	fixedcapvec<fixed32, WORKSPACE_SIZE + 1> y;
	fixedcapvec<fixed32, WORKSPACE_SIZE + 1> z;
	fixedcapvec<float, WORKSPACE_SIZE + 1> h;
	fixedcapvec<char, WORKSPACE_SIZE + 1> rungs;
};

struct hydro_record1 {
	fixed32 x;
	fixed32 y;
	fixed32 z;
	float h;
	char rung;
};

struct hydro_record2 {
	float gamma;
	float vx;
	float vy;
	float vz;
	float ent;
	float alpha;
	float f0;
	float fvel;
	float gx;
	float gy;
	float gz;
};

struct dif_record1 {
	fixed32 x;
	fixed32 y;
	fixed32 z;
	float h;
	char rung;
};

struct dif_record2 {
	dif_vector vec;
	float gamma;
	float difco;
	float kappa;
	float mmw;
	char oldrung;
};

struct hydro_workspace {
	fixedcapvec<hydro_record1, WORKSPACE_SIZE + 2> rec1_main;
	fixedcapvec<hydro_record2, WORKSPACE_SIZE + 2> rec2_main;
	fixedcapvec<hydro_record1, HYDRO_SIZE + 2> rec1;
	fixedcapvec<hydro_record2, HYDRO_SIZE + 2> rec2;
};

struct dif_workspace {
	fixedcapvec<dif_record1, WORKSPACE_SIZE + 2> rec1_main;
	fixedcapvec<dif_record2, WORKSPACE_SIZE + 2> rec2_main;
	fixedcapvec<dif_record1, HYDRO_SIZE + 2> rec1;
	fixedcapvec<dif_record2, HYDRO_SIZE + 2> rec2;
};

struct deposit_workspace {
	fixedcapvec<float, WORKSPACE_SIZE + 2> sn;
	fixedcapvec<fixed32, WORKSPACE_SIZE + 2> x;
	fixedcapvec<fixed32, WORKSPACE_SIZE + 2> y;
	fixedcapvec<fixed32, WORKSPACE_SIZE + 2> z;
	fixedcapvec<float, WORKSPACE_SIZE + 2> vx;
	fixedcapvec<float, WORKSPACE_SIZE + 2> vy;
	fixedcapvec<float, WORKSPACE_SIZE + 2> vz;
	fixedcapvec<float, WORKSPACE_SIZE + 2> h;
};

struct courant_record1 {
	fixed32 x;
	fixed32 y;
	fixed32 z;
	float h;
};

struct courant_record2 {
//	float Y;
//	float Z;
	float gamma;
	float vx;
	float vy;
	float vz;
	float gx;
	float gy;
	float gz;
	float ent;
	float T;
	float lambda_e;
	float mmw;
	float alpha;
};

struct courant_workspace {
	fixedcapvec<courant_record1, WORKSPACE_SIZE + 3> rec1_main;
	fixedcapvec<courant_record2, WORKSPACE_SIZE + 3> rec2_main;
	fixedcapvec<courant_record1, HYDRO_SIZE + 3> rec1;
	fixedcapvec<courant_record2, HYDRO_SIZE + 3> rec2;
};

#define SMOOTHLEN_BLOCK_SIZE 512
#define HYDRO_BLOCK_SIZE 32

struct sph_reduction {
	int counter;
	int flag;
	float hmin;
	float hmax;
	float vsig_max;
	double flops;
	int max_rung_hydro;
	int max_rung_grav;
	int max_rung;
};

__global__ void sph_cuda_smoothlen(sph_run_params params, sph_run_cuda_data data, smoothlen_workspace* workspaces, sph_reduction* reduce) {
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int block_size = blockDim.x;
	__shared__
	int index;
	__shared__
	float error;
	smoothlen_workspace& ws = workspaces[bid];
	if (tid == 0) {
		index = atomicAdd(&reduce->counter, 1);
	}
	__syncthreads();
	array<fixed32, NDIM> x;
	while (index < data.nselfs) {

		int flops = 0;
		ws.x.resize(0);
		ws.y.resize(0);
		ws.z.resize(0);
		const sph_tree_node& self = data.trees[data.selfs[index]];
		//	PRINT( "%i\n", self.neighbor_range.second - self.neighbor_range.first);
		for (int ni = self.neighbor_range.first; ni < self.neighbor_range.second; ni++) {
			//PRINT( "%i\n", -self.neighbor_range.first+self.neighbor_range.second);
			const sph_tree_node& other = data.trees[data.neighbors[ni]];
			const int maxpi = round_up(other.part_range.second - other.part_range.first, block_size) + other.part_range.first;
			for (int pi = other.part_range.first + tid; pi < maxpi; pi += block_size) {
				bool contains = false;
				int j;
				int total;
				if (pi < other.part_range.second) {
					x[XDIM] = data.x[pi];
					x[YDIM] = data.y[pi];
					x[ZDIM] = data.z[pi];
					if (self.outer_box.contains(x)) {
						contains = true;
					}
				}
				j = contains;
				compute_indices<SMOOTHLEN_BLOCK_SIZE>(j, total);
				const int offset = ws.x.size();
				const int next_size = offset + total;
				ws.x.resize(next_size);
				ws.y.resize(next_size);
				ws.z.resize(next_size);
				if (contains) {
					const int k = offset + j;
					ws.x[k] = x[XDIM];
					ws.y[k] = x[YDIM];
					ws.z[k] = x[ZDIM];
				}
			}
		}

		float hmin = 1e+20;
		float hmax = 0.0;
		for (int i = self.part_range.first; i < self.part_range.second; i++) {
			if (data.rungs[i] >= params.min_rung) {
				const int snki = self.sink_part_range.first - self.part_range.first + i;
				x[XDIM] = data.x[i];
				x[YDIM] = data.y[i];
				x[ZDIM] = data.z[i];
				int count;
				float f;
				float dfdh;
				int box_xceeded = false;
				int iter = 0;
				float dh;
				float& h = data.h_snk[snki];
				do {
					float max_dh = h / sqrtf(iter + 100);
					const float hinv = 1.f / h; // 4
					const float h2 = sqr(h);    // 1
					count = 0;
					f = 0.f;
					dfdh = 0.f;
					for (int j = tid; j < ws.x.size(); j += block_size) {
						const float dx = distance(x[XDIM], ws.x[j]); // 2
						const float dy = distance(x[YDIM], ws.y[j]); // 2
						const float dz = distance(x[ZDIM], ws.z[j]); // 2
						const float r2 = sqr(dx, dy, dz);            // 2
						const float r = sqrt(r2);                    // 4
						const float q = r * hinv;                    // 1
						flops += 15;
						if (q < 1.f) {                               // 1
							const float w = kernelW(q); // 4
							const float dwdh = -q * dkernelW_dq(q) * hinv; // 3
							f += w;                                   // 1
							dfdh += dwdh;                             // 1
							flops += 15;
							count++;
						}
					}
					shared_reduce_add<float, SMOOTHLEN_BLOCK_SIZE>(f);
					shared_reduce_add<int, SMOOTHLEN_BLOCK_SIZE>(count);
					shared_reduce_add<float, SMOOTHLEN_BLOCK_SIZE>(dfdh);
					dh = 0.2f * h;
					if (count > 1) {
						f -= data.N * float(3.0 / (4.0 * M_PI));
						dh = -f / dfdh;
						dh = fminf(fmaxf(dh, -max_dh), max_dh);
					}
					error = fabsf(f) / (data.N * float(3.0 / (4.0 * M_PI)));
					__syncthreads();
					if (tid == 0) {
						h += dh;
						if (iter > 30) {
							PRINT("over iteration on h solve - %i %e %e %e %e %i\n", iter, h, dh, max_dh, error, count);
						}
					}
					__syncthreads();
					for (int dim = 0; dim < NDIM; dim++) {
						if (distance(self.outer_box.end[dim], x[dim]) - h < 0.0f) {
							box_xceeded = true;
							break;
						}
						if (distance(x[dim], self.outer_box.begin[dim]) - h < 0.0f) {
							box_xceeded = true;
							break;
						}
					}
					iter++;
					if (max_dh / h < SPH_SMOOTHLEN_TOLER) {
						if (tid == 0) {
							PRINT("density solver failed to converge %i\n", ws.x.size());
							__trap();
						}
					}
					shared_reduce_add<int, SMOOTHLEN_BLOCK_SIZE>(box_xceeded);
				} while (error > SPH_SMOOTHLEN_TOLER && !box_xceeded);
				if (tid == 0 && h <= 0.f) {
					PRINT("Less than ZERO H! sph.cu %e\n", h);
					__trap();
				}
				//	if (tid == 0)
				//	PRINT("%i %e\n", count, data.N);
				//		PRINT( "%e\n", h);
				hmin = fminf(hmin, h);
				hmax = fmaxf(hmax, h);
				if (tid == 0) {
					if (box_xceeded) {
						atomicAdd(&reduce->flag, 1);
					}
				}
			}
		}
		shared_reduce_add<int, SMOOTHLEN_BLOCK_SIZE>(flops);
		if (tid == 0) {
			atomicAdd(&reduce->flops, (double) flops);
			atomicMax(&reduce->hmax, hmax);
			atomicMin(&reduce->hmin, hmin);
			index = atomicAdd(&reduce->counter, 1);
		}
		__syncthreads();
	}

}

__global__ void sph_cuda_mark_semiactive(sph_run_params params, sph_run_cuda_data data, mark_semiactive_workspace* workspaces, sph_reduction* reduce) {
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int block_size = blockDim.x;
	__shared__
	int index;
	mark_semiactive_workspace& ws = workspaces[bid];
	if (tid == 0) {
		index = atomicAdd(&reduce->counter, 1);
	}
	__syncthreads();
	array<fixed32, NDIM> x;
	while (index < data.nselfs) {
		int flops = 0;
		ws.x.resize(0);
		ws.y.resize(0);
		ws.z.resize(0);
		ws.h.resize(0);
		ws.rungs.resize(0);
		const sph_tree_node& self = data.trees[data.selfs[index]];
		//	PRINT( "%i\n", self.neighbor_range.second - self.neighbor_range.first);
		for (int ni = self.neighbor_range.first; ni < self.neighbor_range.second; ni++) {
			//PRINT( "%i\n", -self.neighbor_range.first+self.neighbor_range.second);
			const sph_tree_node& other = data.trees[data.neighbors[ni]];
			const int maxpi = round_up(other.part_range.second - other.part_range.first, block_size) + other.part_range.first;
			for (int pi = other.part_range.first + tid; pi < maxpi; pi += block_size) {
				bool contains = false;
				int j;
				int total;
				float h;
				int rung;
				if (pi < other.part_range.second) {
					h = data.h[pi];
					rung = data.rungs[pi];
					if (rung >= params.min_rung) {
						x[XDIM] = data.x[pi];
						x[YDIM] = data.y[pi];
						x[ZDIM] = data.z[pi];
						if (self.outer_box.contains(x)) {
							contains = true;
						}
						if (!contains) {
							contains = true;
							for (int dim = 0; dim < NDIM; dim++) {
								if (distance(x[dim], self.inner_box.begin[dim]) + h < 0.f) {
									contains = false;
									break;
								}
								if (distance(self.inner_box.end[dim], x[dim]) + h < 0.f) {
									contains = false;
									break;
								}
							}
						}
					}
				}
				j = contains;
				compute_indices<SMOOTHLEN_BLOCK_SIZE>(j, total);
				const int offset = ws.x.size();
				const int next_size = offset + total;
				ws.x.resize(next_size);
				ws.y.resize(next_size);
				ws.z.resize(next_size);
				ws.h.resize(next_size);
				ws.rungs.resize(next_size);
				if (contains) {
					if (j >= total) {
						PRINT("%i %i\n", j, total);
					}
					ASSERT(j < total);
					const int k = offset + j;
					ASSERT(k < next_size);
					ASSERT(k < ws.x.size());
					ASSERT(k < ws.y.size());
					ASSERT(k < ws.z.size());
					ASSERT(k < ws.h.size());
					ws.x[k] = x[XDIM];
					ws.y[k] = x[YDIM];
					ws.z[k] = x[ZDIM];
					ws.h[k] = h;
					ws.rungs[k] = rung;
				}
			}
		}

		for (int i = self.part_range.first; i < self.part_range.second; i++) {
			const int snki = self.sink_part_range.first - self.part_range.first + i;
			if (data.rungs[i] >= params.min_rung) {
				if (tid == 0) {
					data.sa_snk[snki] = true;
				}
			} else {
				const auto x0 = data.x[i];
				const auto y0 = data.y[i];
				const auto z0 = data.z[i];
				const auto h0 = data.h[i];
				const auto h02 = sqr(h0);
				int semiactive = 0;
				const int jmax = round_up(ws.x.size(), block_size);
				if (tid == 0) {
					data.sa_snk[snki] = false;
				}
				for (int j = tid; j < jmax; j += block_size) {
					if (j < ws.x.size()) {
						const auto x1 = ws.x[j];
						const auto y1 = ws.y[j];
						const auto z1 = ws.z[j];
						const auto h1 = ws.h[j];
						const auto h12 = sqr(h1);
						const float dx = distance(x0, x1);
						const float dy = distance(y0, y1);
						const float dz = distance(z0, z1);
						const float r2 = sqr(dx, dy, dz);
						if (r2 < fmaxf(h02, h12)) {
							//			PRINT( "SEMIACTIVE\n");
							semiactive++;
						}
					}
					shared_reduce_add<int>(semiactive);
					if (semiactive) {
						if (tid == 0) {
							data.sa_snk[snki] = true;
						}
						break;
					}
				}
			}
		}
		shared_reduce_add<int, SMOOTHLEN_BLOCK_SIZE>(flops);
		if (tid == 0) {
			index = atomicAdd(&reduce->counter, 1);
		}
		__syncthreads();
	}

}

__global__ void sph_cuda_diffusion(sph_run_params params, sph_run_cuda_data data, dif_workspace* workspaces, sph_reduction* reduce) {
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int block_size = blockDim.x;
	__shared__
	int index;
	dif_workspace& ws = workspaces[bid];
	if (tid == 0) {
		index = atomicAdd(&reduce->counter, 1);
	}
	__syncthreads();
	array<fixed32, NDIM> x;
	int flops = 0;
	while (index < data.nselfs) {
		const sph_tree_node& self = data.trees[data.selfs[index]];
		ws.rec1_main.resize(0);
		ws.rec2_main.resize(0);
		for (int ni = self.neighbor_range.first; ni < self.neighbor_range.second; ni++) {
			const sph_tree_node& other = data.trees[data.neighbors[ni]];
			const int maxpi = round_up(other.part_range.second - other.part_range.first, block_size) + other.part_range.first;
			for (int pi = other.part_range.first + tid; pi < maxpi; pi += block_size) {
				bool contains = false;
				int j;
				int total;
				if (pi < other.part_range.second) {
					x[XDIM] = data.x[pi];
					x[YDIM] = data.y[pi];
					x[ZDIM] = data.z[pi];
					if (self.outer_box.contains(x)) {											// 24
						contains = true;
					}
					flops += 24;
					if (!contains) {
						contains = true;
						const float& h = data.h[pi];
						for (int dim = 0; dim < NDIM; dim++) {
							if (distance(x[dim], self.inner_box.begin[dim]) + h < 0.f) { // 4
								contains = false;
								break;
							}
							if (distance(self.inner_box.end[dim], x[dim]) + h < 0.f) {   // 4
								contains = false;
								break;
							}
						}
						flops += 24;
					}
				}
				j = contains;
				compute_indices<HYDRO_BLOCK_SIZE>(j, total);
				const int offset = ws.rec1_main.size();
				const int next_size = offset + total;
				ws.rec1_main.resize(next_size);
				ws.rec2_main.resize(next_size);
				if (contains) {
					const int k = offset + j;
					ws.rec1_main[k].x = x[XDIM];
					ws.rec1_main[k].y = x[YDIM];
					ws.rec1_main[k].z = x[ZDIM];
					ws.rec1_main[k].h = data.h[pi];
					ws.rec1_main[k].rung = data.rungs[pi];
					ws.rec2_main[k].difco = data.difco[pi];
					ws.rec2_main[k].kappa = data.kappa[pi];
					ws.rec2_main[k].gamma = data.gamma[pi];
					ws.rec2_main[k].vec = data.dif_vec[pi];
					ws.rec2_main[k].oldrung = data.oldrung[pi];
					ws.rec2_main[k].mmw = data.mmw[pi];
				}
			}
		}
		for (int i = self.part_range.first; i < self.part_range.second; i++) {
			int myrung = data.rungs[i];
			const int snki = self.sink_part_range.first - self.part_range.first + i;
			bool active = myrung >= params.min_rung;
			bool semi_active = !active && data.sa_snk[snki];
			bool use = active || semi_active;
			const float m = data.m;
			const float minv = 1.f / m;																	// 4
			const float c0 = float(3.0f / 4.0f / M_PI * data.N);
			const float c0inv = float(1.0f / c0);

			if (use) {
				const auto x_i = data.x[i];
				const auto y_i = data.y[i];
				const auto z_i = data.z[i];
				const float h_i = data.h[i];
				const float hinv_i = 1.f / h_i;															// 4
				const float h3inv_i = (sqr(hinv_i) * hinv_i);										// 6
				const float rho_i = m * c0 * h3inv_i;													// 2
				const float rhoinv_i = minv * c0inv * sqr(h_i) * h_i;								// 5
				const float difco_i = data.difco[i];
				const float kappa_i = data.kappa[i];
				const auto vec0_i = data.vec0_snk[snki];
				const auto vec_i = data.dif_vec[i];
				const auto gamma_i = data.gamma[i];
				const auto mmw_i = data.mmw[i];
				const int jmax = round_up(ws.rec1_main.size(), block_size);
				ws.rec1.resize(0);
				ws.rec2.resize(0);
				for (int j = tid; j < jmax; j += block_size) {
					bool flag = false;
					int k;
					int total;
					if (j < ws.rec1_main.size()) {
						const auto rec = ws.rec1_main[j];
						const auto x_j = rec.x;
						const auto y_j = rec.y;
						const auto z_j = rec.z;
						const float h_j = rec.h;
						const float dx = distance(x_i, x_j);									// 2
						const float dy = distance(y_i, y_j);									// 2
						const float dz = distance(z_i, z_j);                         	// 2
						const float h2max = sqr(fmaxf(h_i, h_j));
						const float r2 = sqr(dx, dy, dz);
						if (r2 < h2max) {
							if (semi_active) {
								if (rec.rung >= params.min_rung) {
									flag = true;
								}
							} else {
								flag = true;
							}
						}
					}
					k = flag;
					compute_indices<HYDRO_BLOCK_SIZE>(k, total);
					const int offset = ws.rec1.size();
					const int next_size = offset + total;
					ws.rec1.resize(next_size);
					ws.rec2.resize(next_size);
					if (flag) {
						const int l = offset + k;
						ws.rec1[l] = ws.rec1_main[j];
						ws.rec2[l] = ws.rec2_main[j];
					}
				}
				dif_vector num;
				float den = 0.f;
				float den_A = 0.f;
				for (int fi = 0; fi < DIFCO_COUNT; fi++) {
					num[fi] = 0.f;
				}
				for (int j = tid; j < ws.rec1.size(); j += block_size) {
					auto rec1 = ws.rec1[j];
					auto rec2 = ws.rec2[j];
					const fixed32 x_j = rec1.x;
					const fixed32 y_j = rec1.y;
					const fixed32 z_j = rec1.z;
					const float x_ij = distance(x_i, x_j);				// 2
					const float y_ij = distance(y_i, y_j);				// 2
					const float z_ij = distance(z_i, z_j);				// 2
					const float r2 = sqr(x_ij, y_ij, z_ij);
					const float r = sqrt(r2);
					const float rinv = 1.0f / (1.0f + r);
					const float gamma_j = rec2.gamma;
					const float kappa_j = rec2.kappa;
					const float difco_j = rec2.difco;
					const float mmw_j = rec2.mmw;
					const float h_j = rec1.h;
					const float hinv_j = 1.f / h_j;															// 4
					const float h3inv_j = sqr(hinv_j) * hinv_j;
					const float rho_j = m * c0 * h3inv_j;													// 2
					const float dt_ij = 0.5f * fminf(rung_dt[myrung], rung_dt[rec1.rung]) * params.t0;
					const float rho_ij = 0.5f * (rho_i + rho_j);
					const float h_ij = 0.5f * (h_i + h_j);
					const float kappa_ij = 2.f * kappa_i * kappa_j / (kappa_i + kappa_j + 1e-30) ;
					const float difco_ij = 2.f * (difco_i * difco_j) / (difco_i + difco_j + 1e-30);
					const float dWdr_ij = 0.5f * (dkernelW_dq(fminf(r * hinv_i, 1.f)) / sqr(sqr(h_i)) + dkernelW_dq(fminf(r * hinv_j, 1.f)) / sqr(sqr(hinv_j)));
					const float diff_factor = -2.f * dt_ij * m / rho_ij * difco_ij * dWdr_ij * rinv;
					const float cond_factor = -dt_ij * m / (rho_i * rho_j) * kappa_ij * dWdr_ij * rinv;
					for (int fi = 0; fi < DIFCO_COUNT; fi++) {
						num[fi] += diff_factor * rec2.vec[fi];
					}
					den += diff_factor;
					float adjust = powf(rho_j, gamma_j - 1.f) * powf(rho_i, 1.f - gamma_i) * mmw_j / mmw_i;
					num[NCHEMFRACS] += cond_factor * rec2.vec[NCHEMFRACS] * adjust;
					den_A += diff_factor + cond_factor;
				}
				for (int fi = 0; fi < DIFCO_COUNT; fi++) {
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(num[fi]);
				}
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(den);
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(den_A);
				if (tid == 0) {
					den += 1.0f;
					den_A += 1.0f;
					num[NCHEMFRACS] += vec0_i[NCHEMFRACS];
					data.dvec_snk[snki][NCHEMFRACS] = num[NCHEMFRACS] / den_A - vec_i[NCHEMFRACS];
					for (int fi = 0; fi < NCHEMFRACS; fi++) {
						num[fi] += vec0_i[fi];
						data.dvec_snk[snki][fi] = num[fi] / den - vec_i[fi];
					}
				}
			}
		}
		shared_reduce_add<int, HYDRO_BLOCK_SIZE>(flops);
		if (tid == 0) {
			atomicAdd(&reduce->flops, (double) flops);
			index = atomicAdd(&reduce->counter, 1);
		}
		flops = 0;
		__syncthreads();
	}
}

#define SIGMA 2.0f
#define ETA1 0.01f
#define ETA2 0.0001f

__global__ void sph_cuda_hydro(sph_run_params params, sph_run_cuda_data data, hydro_workspace* workspaces, sph_reduction* reduce) {
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int block_size = blockDim.x;
	__shared__
	int index;
	hydro_workspace& ws = workspaces[bid];
	if (tid == 0) {
		index = atomicAdd(&reduce->counter, 1);
	}
	__syncthreads();
	array<fixed32, NDIM> x;
	int flops = 0;
	while (index < data.nselfs) {
		const sph_tree_node& self = data.trees[data.selfs[index]];
		ws.rec1_main.resize(0);
		ws.rec2_main.resize(0);
		for (int ni = self.neighbor_range.first; ni < self.neighbor_range.second; ni++) {
			const sph_tree_node& other = data.trees[data.neighbors[ni]];
			const int maxpi = round_up(other.part_range.second - other.part_range.first, block_size) + other.part_range.first;
			for (int pi = other.part_range.first + tid; pi < maxpi; pi += block_size) {
				bool contains = false;
				int j;
				int total;
				if (pi < other.part_range.second) {
					x[XDIM] = data.x[pi];
					x[YDIM] = data.y[pi];
					x[ZDIM] = data.z[pi];
					if (self.outer_box.contains(x)) {
						contains = true;
					}
					if (!contains) {
						contains = true;
						const float& h = data.h[pi];
						for (int dim = 0; dim < NDIM; dim++) {
							if (distance(x[dim], self.inner_box.begin[dim]) + h < 0.f) {
								contains = false;
								break;
							}
							if (distance(self.inner_box.end[dim], x[dim]) + h < 0.f) {
								contains = false;
								break;
							}
						}
					}
				}
				j = contains;
				compute_indices<HYDRO_BLOCK_SIZE>(j, total);
				const int offset = ws.rec1_main.size();
				const int next_size = offset + total;
				ws.rec1_main.resize(next_size);
				ws.rec2_main.resize(next_size);
				if (contains) {
					const int k = offset + j;
					ws.rec1_main[k].x = x[XDIM];
					ws.rec1_main[k].y = x[YDIM];
					ws.rec1_main[k].z = x[ZDIM];
					ws.rec1_main[k].h = data.h[pi];
					ws.rec1_main[k].rung = data.rungs[pi];
					if (data.gamma) {
						ws.rec2_main[k].gamma = data.gamma[pi];
					} else {
						ws.rec2_main[k].gamma = 5. / 3.;
					}
					ws.rec2_main[k].vx = data.vx[pi];
					ws.rec2_main[k].vy = data.vy[pi];
					ws.rec2_main[k].vz = data.vz[pi];
					ws.rec2_main[k].gx = data.gx[pi];
					ws.rec2_main[k].gy = data.gy[pi];
					ws.rec2_main[k].gz = data.gz[pi];
					ws.rec2_main[k].ent = data.ent[pi];
					ws.rec2_main[k].f0 = data.f0[pi];
					ws.rec2_main[k].fvel = data.fvel[pi];
					ws.rec2_main[k].alpha = data.alpha[pi];
				}
			}
		}
		for (int i = self.part_range.first; i < self.part_range.second; i++) {
			int myrung = data.rungs[i];
			const int snki = self.sink_part_range.first - self.part_range.first + i;
			bool active = myrung >= params.min_rung;
			bool semi_active = !active && data.sa_snk[snki];
			bool use = active || semi_active;
			bool first_step = params.phase == 1 && active;
			if (first_step && use && tid == 0) {
				data.dent_con[snki] = 0.0f;
				data.dvx_con[snki] = 0.f;
				data.dvy_con[snki] = 0.f;
				data.dvz_con[snki] = 0.f;
			}
			const float m = data.m;
			const float minv = 1.f / m;
			const float c0 = float(3.0f / 4.0f / M_PI * data.N);
			const float c0inv = 1.0f / c0;
			if (use) {
				const fixed32 x_i = data.x[i];
				const fixed32 y_i = data.y[i];
				const fixed32 z_i = data.z[i];
				const float vx_i = data.vx[i];
				const float vy_i = data.vy[i];
				const float vz_i = data.vz[i];
				const float gx_i = data.gx[i];
				const float gy_i = data.gy[i];
				const float gz_i = data.gz[i];
				const float h_i = data.h[i];
				const float hinv_i = 1.f / h_i;															// 4
				const float h3inv_i = sqr(hinv_i) * hinv_i;
				const float rho_i = m * c0 * h3inv_i;													// 2
				const float rhoinv_i = minv * c0inv * sqr(h_i) * h_i;								// 5
				const float ent_i = data.ent[i];
				const float gamma_i = data.gamma[i];
				const float p_i = ent_i * powf(rho_i, gamma_i);								// 5
				const float c_i = sqrtf(gamma_i * p_i * rhoinv_i);									// 6
				const float fvel_i = data.fvel[i];
				const float f0_i = data.f0[i];
				const float alpha_i = data.alpha[i];
				const float rho1mgamma_i = powf(rho_i, 1.0f - gamma_i);						// 5
				const int jmax = round_up(ws.rec1_main.size(), block_size);
				flops += 36;
				ws.rec1.resize(0);
				ws.rec2.resize(0);
				for (int j = tid; j < jmax; j += block_size) {
					bool flag = false;
					int k;
					int total;
					if (j < ws.rec1_main.size()) {
						const auto rec = ws.rec1_main[j];
						const auto x_j = rec.x;
						const auto y_j = rec.y;
						const auto z_j = rec.z;
						const float h_j = rec.h;
						const float dx = distance(x_j, x_i);
						const float dy = distance(y_j, y_i);
						const float dz = distance(z_j, z_i);
						const float h2max = sqr(fmaxf(h_j, h_i));
						const float r2 = sqr(dx, dy, dz);
						if (r2 < h2max) {
							if (semi_active) {
								if (rec.rung >= params.min_rung) {
									flag = true;
								}
							} else {
								flag = true;
							}
						}
					}
					k = flag;
					compute_indices<HYDRO_BLOCK_SIZE>(k, total);
					const int offset = ws.rec1.size();
					const int next_size = offset + total;
					ws.rec1.resize(next_size);
					ws.rec2.resize(next_size);
					if (flag) {
						const int l = offset + k;
						ws.rec1[l] = ws.rec1_main[j];
						ws.rec2[l] = ws.rec2_main[j];
					}
				}
				float dvxdx = 0.f;
				float dvxdy = 0.f;
				float dvxdz = 0.f;
				float dvydx = 0.f;
				float dvydy = 0.f;
				float dvydz = 0.f;
				float dvzdx = 0.f;
				float dvzdy = 0.f;
				float dvzdz = 0.f;
				float ddivv_dt = 0.f;
				float dent_pred = 0.f;
				float dvx_pred = 0.f;
				float dvy_pred = 0.f;
				float dvz_pred = 0.f;
				float dent_con = 0.f;
				float dvx_con = 0.f;
				float dvy_con = 0.f;
				float dvz_con = 0.f;
				float vsig = 0.f;
				const float ainv = 1.0f / params.a;
				for (int j = tid; j < ws.rec1.size(); j += block_size) {
					auto rec1 = ws.rec1[j];
					auto rec2 = ws.rec2[j];
					const float vx_j = rec2.vx;
					const float vy_j = rec2.vy;
					const float vz_j = rec2.vz;
					const float gx_j = rec2.gx;
					const float gy_j = rec2.gy;
					const float gz_j = rec2.gz;
					const fixed32 x_j = rec1.x;
					const fixed32 y_j = rec1.y;
					const fixed32 z_j = rec1.z;
					const float h_j = rec1.h;
					const float hinv_j = 1.f / h_j;															// 4
					const float h3inv_j = sqr(hinv_j) * hinv_j;
					const float rho_j = m * c0 * h3inv_j;													// 2
					const float rhoinv_j = minv * c0inv * sqr(h_j) * h_j;								// 5
					const float ent_j = rec2.ent;
					const float gamma_j = rec2.gamma;
					const float p_j = ent_j * powf(rho_j, gamma_j);								// 5
					const float c_j = sqrtf(gamma_j * p_j * rhoinv_j);									// 6
					const float fvel_j = rec2.fvel;
					const float f0_j = rec2.f0;
					const float alpha_j = rec2.alpha;
					const float x_ij = distance(x_i, x_j);				// 2
					const float y_ij = distance(y_i, y_j);				// 2
					const float z_ij = distance(z_i, z_j);				// 2
					const float vx_ij = vx_i - vx_j;
					const float vy_ij = vy_i - vy_j;
					const float vz_ij = vz_i - vz_j;
					const float r2 = sqr(x_ij, y_ij, z_ij);
					const float r = sqrt(r2);
					const float rinv = 1.0f / (1.0f + r);
					const float alpha_ij = 0.5f * (alpha_i * fvel_i + alpha_j * fvel_j);
					const float h_ij = 0.5f * (h_i + h_j);
					const float vdotr_ij = fminf(0.0f, x_ij * vx_ij + y_ij * vy_ij + z_ij * vz_ij);
					const float u_ij = vdotr_ij * h_ij / (r2 + ETA1 * sqr(h_ij));
					const float c_ij = 0.5f * (c_i + c_j);
					vsig = fmaxf(vsig, c_ij - vdotr_ij * rinv);
					const float rho_ij = 0.5f * (rho_i + rho_j);
					const float Pi = -alpha_ij * u_ij * (c_ij - SPH_BETA * u_ij) / rho_ij;
					const float q_i = fminf(r * hinv_i, 1.f);								// 1
					const float q_j = fminf(r * hinv_j, 1.f);									// 1
					const float dWdr_i = dkernelW_dq(q_i) * hinv_i * h3inv_i;
					const float dWdr_j = dkernelW_dq(q_j) * hinv_j * h3inv_j;
					const float dWdr_ij = 0.5f * (dWdr_i + dWdr_j);
					const float dWdr_x_ij = x_ij * rinv * dWdr_ij;
					const float dWdr_y_ij = y_ij * rinv * dWdr_ij;
					const float dWdr_z_ij = z_ij * rinv * dWdr_ij;
					const float dp_i = f0_i * p_i * powf(rho_i, SIGMA - 2.f) * powf(rho_j, -SIGMA);
					const float dp_j = f0_j * p_j * powf(rho_j, SIGMA - 2.f) * powf(rho_i, -SIGMA);
					const float dvx_dt = -m * (dp_i + dp_j + Pi) * dWdr_x_ij;
					const float dvy_dt = -m * (dp_i + dp_j + Pi) * dWdr_y_ij;
					const float dvz_dt = -m * (dp_i + dp_j + Pi) * dWdr_z_ij;
					const float dWdr_x_i = dWdr_i * rinv * x_ij;
					const float dWdr_y_i = dWdr_i * rinv * y_ij;
					const float dWdr_z_i = dWdr_i * rinv * z_ij;
					const float mrhoinv_i = m * rhoinv_i;
					float tmp = 0.f;
					tmp += (p_i * powf(rho_i, -SIGMA) + p_j * powf(rho_j, -SIGMA)) * (powf(rho_i, SIGMA - 1.0f) - powf(rho_j, SIGMA - 1.f));
					tmp += (powf(rho_i, SIGMA - 2.0f) + powf(rho_j, SIGMA - 2.f)) * (p_i * powf(rho_i, 1.f - SIGMA) - p_j * powf(rho_j, 1.f - SIGMA));
					ddivv_dt += m * tmp * dWdr_ij / rho_ij * rinv;
					ddivv_dt += mrhoinv_i * (gx_j - gx_i) * dWdr_x_i;
					ddivv_dt += mrhoinv_i * (gy_j - gy_i) * dWdr_y_i;
					ddivv_dt += mrhoinv_i * (gz_j - gz_i) * dWdr_z_i;
					dvxdx -= mrhoinv_i * vx_ij * dWdr_x_i;
					dvydx -= mrhoinv_i * vy_ij * dWdr_x_i;
					dvzdx -= mrhoinv_i * vz_ij * dWdr_x_i;
					dvxdy -= mrhoinv_i * vx_ij * dWdr_y_i;
					dvydy -= mrhoinv_i * vy_ij * dWdr_y_i;
					dvzdy -= mrhoinv_i * vz_ij * dWdr_y_i;
					dvxdz -= mrhoinv_i * vx_ij * dWdr_z_i;
					dvydz -= mrhoinv_i * vy_ij * dWdr_z_i;
					dvzdz -= mrhoinv_i * vz_ij * dWdr_z_i;
					float dt_pred, dt_con;
					dt_pred = 0.5f * rung_dt[myrung] * params.t0;		// 2
					dt_con = fminf(0.5f * rung_dt[rec1.rung] * (params.t0), dt_pred); // 3
					const float tmp2 = (vx_ij * dWdr_x_ij + vy_ij * dWdr_y_ij + vz_ij * dWdr_z_ij);
					const float dA_dt = 0.5f * m * (gamma_i - 1.f) * rho1mgamma_i * Pi * tmp2;
					if (first_step) {
						dent_pred += dA_dt * dt_pred;							// 2
						dvx_pred += dvx_dt * dt_pred;							// 2
						dvy_pred += dvy_dt * dt_pred;							// 2
						dvz_pred += dvz_dt * dt_pred;							// 2
						flops += 8;
					}
					dent_con += dA_dt * dt_con;									// 2
					dvx_con += dvx_dt * dt_con;								// 2
					dvy_con += dvy_dt * dt_con;								// 2
					dvz_con += dvz_dt * dt_con;								// 2
					flops += 181;
				}
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(dent_con);
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(dvx_con);
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(dvy_con);
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(dvz_con);
				if (first_step) {
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(dent_pred);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(dvx_pred);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(dvy_pred);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(dvz_pred);
				}
				float divv = f0_i * (dvxdx + dvydy + dvzdz);
				float curlv_x = f0_i * (dvzdy - dvydz);
				float curlv_y = f0_i * (-dvzdx + dvxdz);
				float curlv_z = f0_i * (dvydx - dvxdy);
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(ddivv_dt);
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(divv);
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(curlv_x);
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(curlv_y);
				shared_reduce_add<float, HYDRO_BLOCK_SIZE>(curlv_z);
				shared_reduce_max<float, HYDRO_BLOCK_SIZE>(vsig);
				if (tid == 0) {
					if (first_step) {
						data.dent_pred[snki] = dent_pred;
						data.dvx_pred[snki] = dvx_pred;
						data.dvy_pred[snki] = dvy_pred;
						data.dvz_pred[snki] = dvz_pred;
					}
					data.dent_con[snki] += dent_con;										// 1
					data.dvx_con[snki] += dvx_con;										// 1
					data.dvy_con[snki] += dvy_con;										// 1
					data.dvz_con[snki] += dvz_con;										// 1
					flops += 4;
					if (params.phase == 1 && !semi_active) {
						data.divv_snk[snki] = divv;
					}
					const float alpha_n = data.alpha_snk[snki];
					float& alpha_np1 = data.alpha_snk[snki];
					const float t0 = h_i / vsig / SPH_VISC_DECAY;
					const float balsara = fabsf(divv) / (sqrt(sqr(curlv_x, curlv_y, curlv_z)) + fabsf(divv) + ETA2 * c_i * hinv_i);
					float S = fmaxf(0.f, -ddivv_dt) * balsara;
					const float alpha_targ = SPH_ALPHA1 / (1.f + sqr(vsig) / (sqr(h_i) * S));
					float dt = 0.5f * rung_dt[myrung] * params.t0; // 3
					const float num = alpha_n + dt / t0 * alpha_targ;
					const float den = 1.f + dt / t0;
					alpha_np1 = fmaxf(alpha_targ, num / den);
				}
			}
		}
		shared_reduce_add<int, HYDRO_BLOCK_SIZE>(flops);
		if (tid == 0) {
			atomicAdd(&reduce->flops, (double) flops);
			index = atomicAdd(&reduce->counter, 1);
		}
		flops = 0;
		__syncthreads();
	}
}

__global__ void sph_cuda_courant(sph_run_params params, sph_run_cuda_data data, courant_workspace* workspaces, sph_reduction* reduce) {
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int block_size = blockDim.x;
	__shared__
	int index;
	courant_workspace& ws = workspaces[bid];
	if (tid == 0) {
		index = atomicAdd(&reduce->counter, 1);
	}
	__syncthreads();
	array<fixed32, NDIM> x;
	float total_vsig_max = 0.;
	int max_rung_hydro = 0;
	int max_rung_grav = 0;
	int max_rung = 0;
	const bool stars = data.gx;
	const float Ginv = 1.f / data.G;
	int flops = 0;

	while (index < data.nselfs) {
		const sph_tree_node& self = data.trees[data.selfs[index]];
		if (self.nactive > 0) {
			ws.rec1_main.resize(0);
			ws.rec2_main.resize(0);
			for (int ni = self.neighbor_range.first; ni < self.neighbor_range.second; ni++) {
				const sph_tree_node& other = data.trees[data.neighbors[ni]];
				const int maxpi = round_up(other.part_range.second - other.part_range.first, block_size) + other.part_range.first;
				for (int pi = other.part_range.first + tid; pi < maxpi; pi += block_size) {
					bool contains = false;
					int j;
					int total;
					if (pi < other.part_range.second) {
						x[XDIM] = data.x[pi];
						x[YDIM] = data.y[pi];
						x[ZDIM] = data.z[pi];
						if (self.outer_box.contains(x)) {
							contains = true;
						}
					}
					j = contains;
					compute_indices<HYDRO_BLOCK_SIZE>(j, total);
					const int offset = ws.rec1_main.size();
					const int next_size = offset + total;
					ws.rec1_main.resize(next_size);
					ws.rec2_main.resize(next_size);
					if (contains) {
						const int k = offset + j;
						ws.rec1_main[k].x = x[XDIM];
						ws.rec1_main[k].y = x[YDIM];
						ws.rec1_main[k].z = x[ZDIM];
						ws.rec2_main[k].vx = data.vx[pi];
						ws.rec2_main[k].vy = data.vy[pi];
						ws.rec2_main[k].vz = data.vz[pi];
						ws.rec2_main[k].ent = data.ent[pi];
						ws.rec1_main[k].h = data.h[pi];
						ws.rec2_main[k].T = data.T[pi];
						ws.rec2_main[k].lambda_e = data.lambda_e[pi];
						ws.rec2_main[k].mmw = data.mmw[pi];
						ws.rec2_main[k].alpha = data.alpha[pi];
						if (data.gamma) {
							ws.rec2_main[k].gamma = data.gamma[pi];
						}
						if (stars) {
							ws.rec2_main[k].gx = data.gx[pi];
							ws.rec2_main[k].gy = data.gy[pi];
							ws.rec2_main[k].gz = data.gz[pi];
						}
					}
				}
			}
			for (int i = self.part_range.first; i < self.part_range.second; i++) {
				int myrung = data.rungs[i];
				bool use = myrung >= params.min_rung;
				const int snki = self.sink_part_range.first - self.part_range.first + i;
				const float m = data.m;
				const float minv = 1.f / m;
				const float c0 = float(3.0f / 4.0f / M_PI * data.N);
				const float c0inv = 1.0f / c0;
				if (use) {
					const auto x_i = data.x[i];
					const auto y_i = data.y[i];
					const auto z_i = data.z[i];
					const auto vx_i = data.vx[i];
					const auto vy_i = data.vy[i];
					const auto vz_i = data.vz[i];
					const float T_i = data.T[i];
					const float h_i = data.h[i];
					const float h2_i = sqr(h_i);
					const float hinv_i = 1.f / h_i;
					const float h3inv_i = sqr(hinv_i) * hinv_i;
					const float rho_i = m * c0 * h3inv_i;
					const float rhoinv_i = minv * c0inv * sqr(h_i) * h_i;
					const float alpha_i = data.alpha[i];
					const float gamma_i = data.gamma[i];
					const float ent_i = data.ent[i];
					const float p_i = ent_i * powf(rho_i, gamma_i);
					const float c_i = sqrtf(gamma_i * p_i * rhoinv_i);
					const float gx_i = data.gx[i];
					const float gy_i = data.gy[i];
					const float gz_i = data.gz[i];
					const int jmax = round_up(ws.rec1_main.size(), block_size);
					ws.rec1.resize(0);
					ws.rec2.resize(0);
					for (int j = tid; j < jmax; j += block_size) {
						bool flag = false;
						int k;
						int total;
						if (j < ws.rec1_main.size()) {
							const auto rec = ws.rec1_main[j];
							const auto x_j = rec.x;
							const auto y_j = rec.y;
							const auto z_j = rec.z;
							const float h_j = rec.h;
							const float dx = distance(x_i, x_j);
							const float dy = distance(y_i, y_j);
							const float dz = distance(z_i, z_j);
							const float r2 = sqr(dx, dy, dz);
							if (r2 < fmaxf(h_i, h_j)) {
								flag = true;
							}
						}
						k = flag;
						compute_indices<HYDRO_BLOCK_SIZE>(k, total);
						const int offset = ws.rec1.size();
						const int next_size = offset + total;
						ws.rec1.resize(next_size);
						ws.rec2.resize(next_size);
						if (flag) {
							const int l = offset + k;
							ws.rec1[l] = ws.rec1_main[j];
							ws.rec2[l] = ws.rec2_main[j];
						}
					}
					float vsig_max = 0.f;
					float dvx_dx = 0.0f;
					float dvx_dy = 0.0f;
					float dvx_dz = 0.0f;
					float dvy_dx = 0.0f;
					float dvy_dy = 0.0f;
					float dvy_dz = 0.0f;
					float dvz_dx = 0.0f;
					float dvz_dy = 0.0f;
					float dvz_dz = 0.0f;
					float drho_dh = 0.f;
					float dgx_dx = 0.f;
					float dgy_dy = 0.f;
					float dgz_dz = 0.f;
					float dT_dx = 0.f;
					float dT_dy = 0.f;
					float dT_dz = 0.f;
					float ax = 0.f;
					float ay = 0.f;
					float az = 0.f;
					for (int j = tid; j < ws.rec1.size(); j += block_size) {
						const auto rec1 = ws.rec1[j];
						const auto rec2 = ws.rec2[j];

						const auto x_j = rec1.x;
						const auto y_j = rec1.y;
						const auto z_j = rec1.z;
						const auto vx_j = rec2.vx;
						const auto vy_j = rec2.vy;
						const auto vz_j = rec2.vz;
//						const auto gx_j = rec2.gx;
//						const auto gy_j = rec2.gy;
//						const auto gz_j = rec2.gz;
						const float T_j = rec2.T;
						const float h_j = rec1.h;
						const float h2_j = sqr(h_j);
						const float hinv_j = 1.f / h_j;
						const float h3inv_j = sqr(hinv_j) * hinv_j;
						const float rho_j = m * c0 * h3inv_j;
						const float rhoinv_j = minv * c0inv * sqr(h_j) * h_j;
						const float alpha_j = rec2.alpha;
						const float gamma_j = rec2.gamma;
						const float ent_j = rec2.ent;
						const float p_j = ent_j * powf(rho_j, gamma_j);
						const float c_j = sqrtf(gamma_j * p_j * rhoinv_j);

						const float x_ij = distance(x_i, x_j);				// 2
						const float y_ij = distance(y_i, y_j);				// 2
						const float z_ij = distance(z_i, z_j);				// 2
						const float vx_ij = vx_i - vx_j;
						const float vy_ij = vy_i - vy_j;
						const float vz_ij = vz_i - vz_j;
						const float r2 = sqr(x_ij, y_ij, z_ij);
						const float r = sqrt(r2);
						const float rinv = 1.0f / (1.0f + r);
						const float alpha_ij = 0.5f * (alpha_i + alpha_j);
						const float h_ij = 0.5f * (h_i + h_j);
						const float vdotr_ij = fminf(0.0f, x_ij * vx_ij + y_ij * vy_ij + z_ij * vz_ij);
						const float u_ij = vdotr_ij * h_ij / (r2 + ETA1 * sqr(h_ij));
						const float c_ij = 0.5f * (c_i + c_j);
						const float rho_ij = 0.5f * (rho_i + rho_j);
						const float Pi = -alpha_ij * u_ij * (c_ij - SPH_BETA * u_ij) / rho_ij;
						const float q_i = fminf(r * hinv_i, 1.f);								// 1
						const float q_j = fminf(r * hinv_j, 1.f);									// 1
						const float dWdr_i = dkernelW_dq(q_i) * hinv_i * h3inv_i;
						const float dWdr_j = dkernelW_dq(q_j) * hinv_j * h3inv_j;
						const float dWdr_ij = 0.5f * (dWdr_i + dWdr_j);
						const float dWdr_x_ij = x_ij * rinv * dWdr_ij;
						const float dWdr_y_ij = y_ij * rinv * dWdr_ij;
						const float dWdr_z_ij = z_ij * rinv * dWdr_ij;
						const float dp_i = p_i * powf(rho_i, SIGMA - 2.f) * powf(rho_j, -SIGMA);
						const float dp_j = p_j * powf(rho_j, SIGMA - 2.f) * powf(rho_i, -SIGMA);
						const float dvx_dt = -m * (dp_i + dp_j + Pi) * dWdr_x_ij;
						const float dvy_dt = -m * (dp_i + dp_j + Pi) * dWdr_y_ij;
						const float dvz_dt = -m * (dp_i + dp_j + Pi) * dWdr_z_ij;
						const float dWdr_x_i = dWdr_i * rinv * x_ij;
						const float dWdr_y_i = dWdr_i * rinv * y_ij;
						const float dWdr_z_i = dWdr_i * rinv * z_ij;
						const float mrhoinv_i = m * rhoinv_i;
						dvx_dx -= mrhoinv_i * vx_ij * dWdr_x_i;
						dvy_dx -= mrhoinv_i * vy_ij * dWdr_x_i;
						dvz_dx -= mrhoinv_i * vz_ij * dWdr_x_i;
						dvx_dy -= mrhoinv_i * vx_ij * dWdr_y_i;
						dvy_dy -= mrhoinv_i * vy_ij * dWdr_y_i;
						dvz_dy -= mrhoinv_i * vz_ij * dWdr_y_i;
						dvx_dz -= mrhoinv_i * vx_ij * dWdr_z_i;
						dvy_dz -= mrhoinv_i * vy_ij * dWdr_z_i;
						dvz_dz -= mrhoinv_i * vz_ij * dWdr_z_i;
						const float hfac = h_i / h_ij;
						float this_vsig = c_ij * hfac;
						if (vdotr_ij < 0.f) {
							this_vsig += 0.6f * alpha_ij * c_ij * hfac;
							this_vsig -= 0.6f * alpha_ij * SPH_BETA * vdotr_ij * hfac;
						}
						vsig_max = fmaxf(vsig_max, this_vsig);									   // 2
						const float W = kernelW(fminf(r * hinv_i, 1.f)) * h3inv_i;      // 14
						const float mrhoinvW = m * rhoinv_i * W;
						ax += dvx_dt;
						ay += dvy_dt;
						az += dvz_dt;
						const float tmp3 = (T_j - T_i) * m * rhoinv_i;
						dT_dx += tmp3 * dWdr_x_i;
						dT_dy += tmp3 * dWdr_y_i;
						dT_dz += tmp3 * dWdr_z_i;
						drho_dh -= (3.f * kernelW(q_i) + dkernelW_dq(q_i));
						/*						if (stars) {
						 dgx_dx += (rec2.gx - mygx) * mydWdr_x * m * myrhoinv;
						 dgy_dy += (rec2.gy - mygy) * mydWdr_y * m * myrhoinv;
						 dgz_dz += (rec2.gz - mygz) * mydWdr_z * m * myrhoinv;
						 }*/

					}
					float div_v = dvx_dx + dvy_dy + dvz_dz;
					float curl_vx = dvz_dy - dvy_dz;
					float curl_vy = -dvz_dx + dvx_dz;
					float curl_vz = dvy_dx - dvx_dy;
					float shear_xx = dvx_dx - (1.f / 3.f) * div_v;
					float shear_yy = dvy_dy - (1.f / 3.f) * div_v;
					float shear_zz = dvz_dz - (1.f / 3.f) * div_v;
					float shear_xy = 0.5f * (dvx_dy + dvy_dx);
					float shear_xz = 0.5f * (dvx_dz + dvz_dx);
					float shear_yz = 0.5f * (dvy_dz + dvz_dy);
					float div_g;
					if (stars) {
						div_g = dgx_dx + dgy_dy + dgz_dz;
						shared_reduce_add<float, HYDRO_BLOCK_SIZE>(div_g);
					}
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(div_g);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(ax);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(ay);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(az);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(dT_dx);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(dT_dy);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(dT_dz);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(shear_xx);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(shear_xy);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(shear_xz);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(shear_yy);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(shear_yz);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(shear_zz);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(div_v);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(curl_vx);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(curl_vy);
					shared_reduce_add<float, HYDRO_BLOCK_SIZE>(curl_vz);
					shared_reduce_max<float, HYDRO_BLOCK_SIZE>(vsig_max);

					if (tid == 0) {
						ax += gx_i;
						ay += gy_i;
						az += gz_i;
						const float sw = ETA2 * c_i * hinv_i;
						const float abs_div_v = fabsf(div_v);
						const float abs_curl_v = sqrtf(sqr(curl_vx, curl_vy, curl_vz));
						const float fvel = abs_div_v / (abs_div_v + abs_curl_v + sw);
						const float c0 = drho_dh * 4.0f * float(M_PI) / (9.0f * data.N);
						const float fpre = 1.0f / (1.0f + c0);
						//	PRINT("%e\n", fpre);
						div_v *= fpre;
						const float dt_cfl = params.a * h_i / vsig_max;
						const float Cdif = SPH_DIFFUSION_C * sqr(h_i) * sqrt(sqr(shear_xx, shear_yy, shear_zz) + 2.f * sqr(shear_xy, shear_xz, shear_yz));
						const float lt = T_i / (sqrt(sqr(dT_dx, dT_dy, dT_dz)) + 1.0e-10f * T_i);
						const float kappa_sp = data.kappa0 / data.colog[i]; // Jubelgas et al 2004, Smith et al 2021
						const float kappa = kappa_sp / (1.f + 4.2f * data.lambda_e[i] / lt);
						const float tmp = data.code_dif_to_cgs * constants::kb / sqr(sqr(params.a));
						float Dcond = 2.f * data.mmw[i] * (data.gamma[i] - 1.f) * kappa / tmp;
						data.kappa_snk[snki] = Dcond;
						data.fvel_snk[snki] = fvel;
						data.f0_snk[snki] = fpre;
						data.difco_snk[snki] = Cdif;
						total_vsig_max = fmaxf(total_vsig_max, vsig_max);
						float dthydro = params.cfl * dt_cfl;
						const float gx = data.gx_snk[snki];
						const float gy = data.gy_snk[snki];
						const float gz = data.gz_snk[snki];
						char& rung = data.rungs[i];
						const float g2 = sqr(gx, gy, gz);
						const float a2 = sqr(ax, ay, az);
						const float hsoft = fminf(fmaxf(h_i, data.hsoft_min), SPH_MAX_SOFT);
						const float factor = data.eta * sqrtf(params.a * hsoft);
						dthydro = fminf(fminf(factor / sqrtf(sqrtf(a2 + 1e-15f)), (float) params.t0), dthydro);
						const float dt_grav = fminf(factor / sqrtf(sqrtf(g2 + 1e-15f)), (float) params.t0);
						const float dt = fminf(dt_grav, dthydro);
						const int rung_hydro = ceilf(log2f(params.t0) - log2f(dthydro));
						const int rung_grav = ceilf(log2f(params.t0) - log2f(dt_grav));
						max_rung_hydro = max(max_rung_hydro, rung_hydro);
						max_rung_grav = max(max_rung_grav, rung_grav);
						rung = max(max((int) max(rung_hydro, rung_grav), max(params.min_rung, (int) rung - 1)), 1);
						max_rung = max(max_rung, rung);
						if (rung < 0 || rung >= MAX_RUNG) {
							if (tid == 0) {
								PRINT("Rung out of range \n");
								__trap();
							}
						}
						if (stars) {
							bool is_eligible = h_i < data.hstar0;
							if (is_eligible) {
								//	PRINT( "Removing sink particle\n");
							}
							/*							bool is_eligible = false;
							 const float N = ws.rec1.size();
							 float tdyn;
							 float mj;
							 float tcool;
							 if (div_v < 0.f) {
							 const float Gn32 = powf(data.G, -1.5);
							 float rho0 = data.rho0_b + data.rho0_c;
							 float delta = -Ginv * float(1.0 / 4.0 / M_PI) * div_g;
							 float delta_b = myrho - data.rho0_b;
							 float rho_tot = (rho0 + delta) * powf(params.a, -3.0);
							 tdyn = sqrtf(3.f * M_PI / (32.f * data.G * rho_tot)) / params.a;
							 if (delta_b / data.rho0_b > 10.0 && delta > 0.f) {
							 tcool = data.tcool_snk[snki];
							 if (tcool < tdyn) {
							 mj = Gn32 * rsqrt(myrho) * sqr(myc) * myc * powf(delta_b / delta, 1.5f) * powf(params.a, -1.5f);
							 const float msph = N * m;
							 if (mj < msph) {
							 is_eligible = true;
							 }
							 }
							 }
							 }*/
							if (is_eligible) {
								//float dt = rung_dt[rung] * params.t0;
								//data.tdyn_snk[snki] = tdyn;
								data.tdyn_snk[snki] = 1e-10f;
							} else {
								data.tdyn_snk[snki] = 1e+38;
							}
						}
					}
				}
			}
			shared_reduce_add<int, HYDRO_BLOCK_SIZE>(flops);
			if (tid == 0) {
				atomicAdd(&reduce->flops, (double) flops);
				index = atomicAdd(&reduce->counter, 1);
			}
			flops = 0;
			__syncthreads();
		}
	}
	if (tid == 0) {
		atomicMax(&reduce->vsig_max, total_vsig_max);
		atomicMax(&reduce->max_rung, max_rung);
		atomicMax(&reduce->max_rung_hydro, max_rung_hydro);
		atomicMax(&reduce->max_rung_grav, max_rung_grav);
	}
}

sph_run_return sph_run_cuda(sph_run_params params, sph_run_cuda_data data, hipStream_t stream) {
	timer tm;
	sph_run_return rc;
	sph_reduction* reduce;
	CUDA_CHECK(hipMallocManaged(&reduce, sizeof(sph_reduction)));
	reduce->counter = reduce->flag = 0;
	reduce->hmin = std::numeric_limits<float>::max();
	reduce->hmax = 0.0f;
	reduce->flops = 0.0;
	reduce->vsig_max = 0.0;
	reduce->max_rung_grav = 0;
	reduce->max_rung_hydro = 0;
	reduce->max_rung = 0;
	static int smoothlen_nblocks;
	static int semiactive_nblocks;
	static int hydro_nblocks;
	static int dif_nblocks;
	static int courant_nblocks;
	static bool first = true;
	static char* workspace_ptr;
	if (first) {
		first = false;
		CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&smoothlen_nblocks, (const void*) sph_cuda_smoothlen, SMOOTHLEN_BLOCK_SIZE, 0));
		smoothlen_nblocks *= cuda_smp_count();
		CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&semiactive_nblocks, (const void*) sph_cuda_mark_semiactive, SMOOTHLEN_BLOCK_SIZE, 0));
		semiactive_nblocks *= cuda_smp_count();
		CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&dif_nblocks, (const void*) sph_cuda_diffusion, HYDRO_BLOCK_SIZE, 0));
		dif_nblocks *= cuda_smp_count();
		CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&hydro_nblocks, (const void*) sph_cuda_hydro, HYDRO_BLOCK_SIZE, 0));
		hydro_nblocks *= cuda_smp_count();
		CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&courant_nblocks, (const void*) sph_cuda_courant, HYDRO_BLOCK_SIZE, 0));
		courant_nblocks *= cuda_smp_count();
		size_t smoothlen_mem = sizeof(smoothlen_workspace) * smoothlen_nblocks;
		size_t semiactive_mem = sizeof(mark_semiactive_workspace) * semiactive_nblocks;
		size_t courant_mem = sizeof(courant_workspace) * courant_nblocks;
		size_t hydro_mem = sizeof(hydro_workspace) * hydro_nblocks;
		size_t dif_mem = sizeof(dif_workspace) * dif_nblocks;
		size_t max_mem = std::max(std::max(std::max(smoothlen_mem, semiactive_mem), std::max(hydro_mem, courant_mem)), dif_mem);
		CUDA_CHECK(hipMalloc(&workspace_ptr, max_mem));
		PRINT("Allocating %i GB in workspace memory\n", max_mem / 1024 / 1024 / 1024);
//		sleep(10);
	}

	switch (params.run_type) {
	case SPH_RUN_SMOOTHLEN: {
		sph_cuda_smoothlen<<<smoothlen_nblocks, SMOOTHLEN_BLOCK_SIZE,0,stream>>>(params,data,(smoothlen_workspace*)workspace_ptr,reduce);
		cuda_stream_synchronize(stream);
		rc.rc = reduce->flag;
		rc.hmin = reduce->hmin;
		rc.hmax = reduce->hmax;
	}
	break;
	case SPH_RUN_MARK_SEMIACTIVE: {
		sph_cuda_mark_semiactive<<<semiactive_nblocks, SMOOTHLEN_BLOCK_SIZE,0,stream>>>(params,data,(mark_semiactive_workspace*)workspace_ptr,reduce);
		cuda_stream_synchronize(stream);
	}
	break;
	case SPH_RUN_HYDRO: {
		timer tm;
		tm.start();
		sph_cuda_hydro<<<hydro_nblocks, HYDRO_BLOCK_SIZE,0,stream>>>(params,data,(hydro_workspace*)workspace_ptr,reduce);
		cuda_stream_synchronize(stream);
		tm.stop();
		auto gflops = reduce->flops / tm.read() / (1024.0*1024*1024);
		PRINT( "HYDRO ran with %e GFLOPs\n", gflops);
	}
	break;
	case SPH_RUN_DIFFUSION: {
		timer tm;
		tm.start();
		sph_cuda_diffusion<<<dif_nblocks, HYDRO_BLOCK_SIZE,0,stream>>>(params,data,(dif_workspace*)workspace_ptr,reduce);
		cuda_stream_synchronize(stream);
		tm.stop();
	}
	break;
	case SPH_RUN_COURANT: {
		sph_cuda_courant<<<courant_nblocks, HYDRO_BLOCK_SIZE,0,stream>>>(params,data,(courant_workspace*)workspace_ptr,reduce);
		cuda_stream_synchronize(stream);
		rc.max_vsig = reduce->vsig_max;
		rc.max_rung_grav = reduce->max_rung_grav;
		rc.max_rung_hydro = reduce->max_rung_hydro;
		rc.max_rung = reduce->max_rung;
	}
}
	CUDA_CHECK(hipFree(reduce));

	return rc;
}
